#include <hip/hip_runtime.h>


#include <cstdint>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <chrono>
#include <vector>
#include <iomanip>

// ──────────────────────────────────────────────────────────────────────
//  Compile-time configuration
// ──────────────────────────────────────────────────────────────────────
constexpr unsigned long long THREAD_SIZE  = 256;
constexpr unsigned long long BLOCK_SIZE   = 1ULL << 23;          //  8 388 608
constexpr unsigned long long BATCH_SIZE   = BLOCK_SIZE * THREAD_SIZE;
constexpr int                RESULTS_BUFFER_SIZE = 8;            //  per batch

// ──────────────────────────────────────────────────────────────────────
//  Simple CUDA error helper
// ──────────────────────────────────────────────────────────────────────
#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t _e = (call);                                            \
        if (_e != hipSuccess) {                                            \
            std::cerr << "CUDA error " << hipGetErrorString(_e)            \
                      << " at " << __FILE__ << ':' << __LINE__ << std::endl;\
            std::exit(EXIT_FAILURE);                                        \
        }                                                                   \
    } while (0)

// ──────────────────────────────────────────────────────────────────────
//  Result container (device + host identical)
// ──────────────────────────────────────────────────────────────────────
struct Result {
    uint64_t guess_bits;   // lower 42 bits hold the actual guess
    uint64_t result_lo;
    uint64_t result_hi;
};

// ──────────────────────────────────────────────────────────────────────
//  Small device helpers
// ──────────────────────────────────────────────────────────────────────
__device__ __forceinline__ uint8_t extractBit(uint64_t v, int idx)
{
    return static_cast<uint8_t>((v >> idx) & 1ULL);
}

__device__ void splitToBits(uint64_t v, uint8_t *bits /* size ≥ 64 */)
{
    #pragma unroll
    for (int i = 0; i < 64; ++i)
        bits[i] = extractBit(v, i);
}

// ──────────────────────────────────────────────────────────────────────
//  0.  Rotate‐left helper
// ──────────────────────────────────────────────────────────────────────
__device__ __forceinline__ uint64_t rotl64(uint64_t x, unsigned r) {
    return (x << r) | (x >> (64u - r));
}

// ──────────────────────────────────────────────────────────────────────
//  5.  Validation
//      Takes the two 64‐bit results and the original seeds,
//      returns true if they round‐trip to nextlong1/nextlong2.
// ──────────────────────────────────────────────────────────────────────
__device__ bool isValid(uint64_t lo, uint64_t hi,
                        uint64_t nextlong1, uint64_t nextlong2)
{
    // compute testOut1 = rotl64(lo+hi,17) + lo
    uint64_t sum0      = lo + hi;
    uint64_t testOut1 = rotl64(sum0, 17) + lo;

    // t = hi ^ lo
    uint64_t t = hi ^ lo;

    // lo' = rotl64(lo,49) ^ t ^ (t << 21)
    uint64_t lo2 = rotl64(lo, 49) ^ t ^ (t << 21);

    // hi' = rotl64(t,28)
    uint64_t hi2 = rotl64(t, 28);

    // testOut2 = rotl64(lo'+hi',17) + lo'
    uint64_t sum1      = lo2 + hi2;
    uint64_t testOut2 = rotl64(sum1, 17) + lo2;

    // only valid if both match
    return (testOut1 == nextlong1 && testOut2 == nextlong2);
}

// ──────────────────────────────────────────────────────────────────────
//  The brute-force kernel
// ──────────────────────────────────────────────────────────────────────
__global__ void bruteKernel(uint64_t nextlong1, uint64_t nextlong2,
                            uint64_t guess_base,
                            Result  *d_results,
                            unsigned int *d_resIndex)
{
    // Global thread index in current batch
    unsigned long long tid = blockIdx.x * blockDim.x + threadIdx.x;

    // The 42-bit value this thread will test
    uint64_t guess = guess_base + tid;          // assumes contiguous space

    // ----------------------------------------------------------------------------
    //  1.   Expand the two 64-bit seeds into bit arrays
    // ----------------------------------------------------------------------------
    
    uint8_t S1_lo[64];
    uint8_t S1_hi[64];
    uint8_t S1_hxl[64];
    uint8_t S1_ca[64];
    uint8_t S1_is[64];
    uint8_t S1_cb[64];
    uint8_t S1_out[64];
    
    uint8_t S2_lo[64];
    uint8_t S2_hi[64];
    uint8_t S2_hxl[64];
    uint8_t S2_ca[64];
    uint8_t S2_is[64];
    uint8_t S2_cb[64];
    uint8_t S2_out[64];
    splitToBits(nextlong1, S1_out);
    splitToBits(nextlong2, S2_out);

    // ----------------------------------------------------------------------------
    //  2.   Expand the 42-bit guess into individual bits
    // ----------------------------------------------------------------------------
    uint8_t guess_bits[64];
    splitToBits(guess, guess_bits);

    // ----------------------------------------------------------------------------
    //  3.   INSERT GENERATED LOGIC HERE
    //       –––––––––––––––––––––––––––
    //  Hundreds of lines like:
    //
    //        uint8_t S1_ca_46 = S1_hxl_47 ^ S1_is_47;
    //        uint8_t S1_ca_47;
    //        uint8_t tmpSum   = carry(S1_ca_46, S1_lo_47, S1_hi_47, &S1_ca_47);
    //
    //  Please paste the auto-generated section replacing the whole placeholder.
    // ----------------------------------------------------------------------------
    //  BEGIN ▼▼▼▼▼ (placeholder block)
    S1_lo[51] = guess_bits[0];
    S1_lo[50] = guess_bits[1];
    S1_lo[49] = guess_bits[2];
    S1_lo[48] = guess_bits[3];
    S1_lo[47] = guess_bits[4];
    S1_lo[46] = guess_bits[5];
    S1_lo[45] = guess_bits[6];
    S1_lo[44] = guess_bits[7];
    S1_lo[43] = guess_bits[8];
    S1_lo[35] = guess_bits[9];
    S1_lo[34] = guess_bits[10];
    S1_lo[33] = guess_bits[11];
    S1_lo[32] = guess_bits[12];
    S1_lo[31] = guess_bits[13];
    S1_lo[30] = guess_bits[14];
    S1_lo[29] = guess_bits[15];
    S1_lo[28] = guess_bits[16];
    S1_lo[27] = guess_bits[17];
    S1_lo[26] = guess_bits[18];
    S1_lo[21] = guess_bits[19];
    S1_lo[17] = guess_bits[20];
    S1_lo[15] = guess_bits[21];
    S1_lo[14] = guess_bits[22];
    S1_lo[13] = guess_bits[23];
    S1_lo[12] = guess_bits[24];
    S1_lo[11] = guess_bits[25];
    S1_lo[9] = guess_bits[26];
    S1_lo[6] = guess_bits[27];
    S1_lo[2] = guess_bits[28];
    S1_lo[1] = guess_bits[29];
    S1_lo[0] = guess_bits[30];
    S1_hxl[47] = guess_bits[31];
    S1_hxl[26] = guess_bits[32];
    S1_hxl[11] = guess_bits[33];
    S1_hxl[7] = guess_bits[34];
    S1_hxl[4] = guess_bits[35];
    S1_is[26] = guess_bits[36];
    S1_is[9] = guess_bits[37];
    S1_is[4] = guess_bits[38];
    S2_is[53] = guess_bits[39];
    S2_is[32] = guess_bits[40];
    S2_is[11] = guess_bits[41];


    S1_hi[47] = S1_lo[47] ^ S1_hxl[47];
    S1_hi[26] = S1_lo[26] ^ S1_hxl[26];
    S1_hi[11] = S1_lo[11] ^ S1_hxl[11];
    S2_lo[11] = S1_lo[26] ^ S1_hxl[11];
    S2_hi[54] = S1_hxl[26];
    S2_hi[39] = S1_hxl[11];
    S2_hi[35] = S1_hxl[7];
    S2_hi[32] = S1_hxl[4];
    S2_hi[11] = S1_hxl[47];
    S2_hxl[11] = S1_lo[26] ^ S1_hxl[47] ^ S1_hxl[11];
    S1_ca[3] = S1_hxl[4] ^ S1_is[4];
    S1_ca[25] = S1_hxl[26] ^ S1_is[26];
    S1_ca[26] = ((S1_ca[25] & S1_lo[26]) | (S1_lo[26] & S1_hi[26]) | (S1_ca[25] & S1_hi[26]));
    S1_is[47] = S1_lo[0] ^ S1_out[0];
    S1_ca[46] = S1_hxl[47] ^ S1_is[47];
    S1_ca[47] = ((S1_ca[46] & S1_lo[47]) | (S1_lo[47] & S1_hi[47]) | (S1_ca[46] & S1_hi[47]));
    S1_cb[0] = S1_is[47] & S1_lo[0];
    S1_is[48] = S1_lo[1] ^ S1_out[1] ^ S1_cb[0];
    S1_hxl[48] = S1_is[48] ^ S1_ca[47];
    S1_hi[48] = S1_lo[48] ^ S1_hxl[48];
    S2_hi[12] = S1_hxl[48];
    S1_ca[48] = ((S1_ca[47] & S1_lo[48]) | (S1_lo[48] & S1_hi[48]) | (S1_ca[47] & S1_hi[48]));
    S1_cb[1] = ((S1_cb[0] & S1_is[48]) | (S1_is[48] & S1_lo[1]) | (S1_cb[0] & S1_lo[1]));
    S1_is[49] = S1_lo[2] ^ S1_out[2] ^ S1_cb[1];
    S1_hxl[49] = S1_is[49] ^ S1_ca[48];
    S1_hi[49] = S1_lo[49] ^ S1_hxl[49];
    S2_hi[13] = S1_hxl[49];
    S1_ca[49] = ((S1_ca[48] & S1_lo[49]) | (S1_lo[49] & S1_hi[49]) | (S1_ca[48] & S1_hi[49]));
    S1_cb[2] = ((S1_cb[1] & S1_is[49]) | (S1_is[49] & S1_lo[2]) | (S1_cb[1] & S1_lo[2]));
    S1_cb[20] = S1_is[4] ^ S1_lo[21] ^ S1_out[21];
    S1_cb[21] = ((S1_cb[20] & S1_is[4]) | (S1_is[4] & S1_lo[21]) | (S1_cb[20] & S1_lo[21]));
    S1_cb[25] = S1_is[9] ^ S1_lo[26] ^ S1_out[26];
    S1_cb[26] = ((S1_cb[25] & S1_is[9]) | (S1_is[9] & S1_lo[26]) | (S1_cb[25] & S1_lo[26]));
    S1_is[10] = S1_lo[27] ^ S1_out[27] ^ S1_cb[26];
    S1_cb[27] = ((S1_cb[26] & S1_is[10]) | (S1_is[10] & S1_lo[27]) | (S1_cb[26] & S1_lo[27]));
    S1_is[11] = S1_lo[28] ^ S1_out[28] ^ S1_cb[27];
    S1_ca[10] = S1_hxl[11] ^ S1_is[11];
    S1_ca[11] = ((S1_ca[10] & S1_lo[11]) | (S1_lo[11] & S1_hi[11]) | (S1_ca[10] & S1_hi[11]));
    S1_cb[28] = ((S1_cb[27] & S1_is[11]) | (S1_is[11] & S1_lo[28]) | (S1_cb[27] & S1_lo[28]));
    S1_is[12] = S1_lo[29] ^ S1_out[29] ^ S1_cb[28];
    S1_hxl[12] = S1_is[12] ^ S1_ca[11];
    S1_hi[12] = S1_lo[12] ^ S1_hxl[12];
    S2_lo[12] = S1_lo[27] ^ S1_hxl[12];
    S2_hi[40] = S1_hxl[12];
    S2_hxl[12] = S1_lo[27] ^ S1_hxl[48] ^ S1_hxl[12];
    S1_ca[12] = ((S1_ca[11] & S1_lo[12]) | (S1_lo[12] & S1_hi[12]) | (S1_ca[11] & S1_hi[12]));
    S1_cb[29] = ((S1_cb[28] & S1_is[12]) | (S1_is[12] & S1_lo[29]) | (S1_cb[28] & S1_lo[29]));
    S1_is[13] = S1_lo[30] ^ S1_out[30] ^ S1_cb[29];
    S1_hxl[13] = S1_is[13] ^ S1_ca[12];
    S1_hi[13] = S1_lo[13] ^ S1_hxl[13];
    S2_lo[13] = S1_lo[28] ^ S1_hxl[13];
    S2_hi[41] = S1_hxl[13];
    S2_hxl[13] = S1_lo[28] ^ S1_hxl[49] ^ S1_hxl[13];
    S1_ca[13] = ((S1_ca[12] & S1_lo[13]) | (S1_lo[13] & S1_hi[13]) | (S1_ca[12] & S1_hi[13]));
    S1_cb[30] = ((S1_cb[29] & S1_is[13]) | (S1_is[13] & S1_lo[30]) | (S1_cb[29] & S1_lo[30]));
    S1_is[14] = S1_lo[31] ^ S1_out[31] ^ S1_cb[30];
    S1_hxl[14] = S1_is[14] ^ S1_ca[13];
    S1_hi[14] = S1_lo[14] ^ S1_hxl[14];
    S2_lo[14] = S1_lo[29] ^ S1_hxl[14];
    S2_hi[42] = S1_hxl[14];
    S1_ca[14] = ((S1_ca[13] & S1_lo[14]) | (S1_lo[14] & S1_hi[14]) | (S1_ca[13] & S1_hi[14]));
    S1_cb[31] = ((S1_cb[30] & S1_is[14]) | (S1_is[14] & S1_lo[31]) | (S1_cb[30] & S1_lo[31]));
    S1_is[15] = S1_lo[32] ^ S1_out[32] ^ S1_cb[31];
    S1_hxl[15] = S1_is[15] ^ S1_ca[14];
    S1_hi[15] = S1_lo[15] ^ S1_hxl[15];
    S2_lo[15] = S1_lo[30] ^ S1_hxl[15];
    S2_hi[43] = S1_hxl[15];
    S1_ca[15] = ((S1_ca[14] & S1_lo[15]) | (S1_lo[15] & S1_hi[15]) | (S1_ca[14] & S1_hi[15]));
    S1_cb[32] = ((S1_cb[31] & S1_is[15]) | (S1_is[15] & S1_lo[32]) | (S1_cb[31] & S1_lo[32]));
    S1_is[16] = S1_lo[33] ^ S1_out[33] ^ S1_cb[32];
    S1_hxl[16] = S1_is[16] ^ S1_ca[15];
    S2_lo[16] = S1_lo[31] ^ S1_hxl[16];
    S2_hi[44] = S1_hxl[16];
    S1_cb[33] = ((S1_cb[32] & S1_is[16]) | (S1_is[16] & S1_lo[33]) | (S1_cb[32] & S1_lo[33]));
    S1_is[17] = S1_lo[34] ^ S1_out[34] ^ S1_cb[33];
    S1_cb[34] = ((S1_cb[33] & S1_is[17]) | (S1_is[17] & S1_lo[34]) | (S1_cb[33] & S1_lo[34]));
    S1_is[18] = S1_lo[35] ^ S1_out[35] ^ S1_cb[34];
    S1_cb[35] = ((S1_cb[34] & S1_is[18]) | (S1_is[18] & S1_lo[35]) | (S1_cb[34] & S1_lo[35]));
    S1_cb[42] = S1_is[26] ^ S1_lo[43] ^ S1_out[43];
    S1_cb[43] = ((S1_cb[42] & S1_is[26]) | (S1_is[26] & S1_lo[43]) | (S1_cb[42] & S1_lo[43]));
    S1_is[27] = S1_lo[44] ^ S1_out[44] ^ S1_cb[43];
    S1_hxl[27] = S1_is[27] ^ S1_ca[26];
    S1_hi[27] = S1_lo[27] ^ S1_hxl[27];
    S2_hi[55] = S1_hxl[27];
    S1_ca[27] = ((S1_ca[26] & S1_lo[27]) | (S1_lo[27] & S1_hi[27]) | (S1_ca[26] & S1_hi[27]));
    S1_cb[44] = ((S1_cb[43] & S1_is[27]) | (S1_is[27] & S1_lo[44]) | (S1_cb[43] & S1_lo[44]));
    S1_is[28] = S1_lo[45] ^ S1_out[45] ^ S1_cb[44];
    S1_hxl[28] = S1_is[28] ^ S1_ca[27];
    S1_hi[28] = S1_lo[28] ^ S1_hxl[28];
    S2_lo[49] = S1_lo[0] ^ S1_hxl[49] ^ S1_hxl[28];
    S2_lo[28] = S1_lo[43] ^ S1_hxl[28] ^ S1_hxl[7];
    S2_hi[56] = S1_hxl[28];
    S1_ca[28] = ((S1_ca[27] & S1_lo[28]) | (S1_lo[28] & S1_hi[28]) | (S1_ca[27] & S1_hi[28]));
    S1_cb[45] = ((S1_cb[44] & S1_is[28]) | (S1_is[28] & S1_lo[45]) | (S1_cb[44] & S1_lo[45]));
    S1_is[29] = S1_lo[46] ^ S1_out[46] ^ S1_cb[45];
    S1_hxl[29] = S1_is[29] ^ S1_ca[28];
    S1_hi[29] = S1_lo[29] ^ S1_hxl[29];
    S2_hi[57] = S1_hxl[29];
    S1_ca[29] = ((S1_ca[28] & S1_lo[29]) | (S1_lo[29] & S1_hi[29]) | (S1_ca[28] & S1_hi[29]));
    S1_cb[46] = ((S1_cb[45] & S1_is[29]) | (S1_is[29] & S1_lo[46]) | (S1_cb[45] & S1_lo[46]));
    S1_is[30] = S1_lo[47] ^ S1_out[47] ^ S1_cb[46];
    S1_hxl[30] = S1_is[30] ^ S1_ca[29];
    S1_hi[30] = S1_lo[30] ^ S1_hxl[30];
    S2_hi[58] = S1_hxl[30];
    S1_ca[30] = ((S1_ca[29] & S1_lo[30]) | (S1_lo[30] & S1_hi[30]) | (S1_ca[29] & S1_hi[30]));
    S1_cb[47] = ((S1_cb[46] & S1_is[30]) | (S1_is[30] & S1_lo[47]) | (S1_cb[46] & S1_lo[47]));
    S1_is[31] = S1_lo[48] ^ S1_out[48] ^ S1_cb[47];
    S1_hxl[31] = S1_is[31] ^ S1_ca[30];
    S1_hi[31] = S1_lo[31] ^ S1_hxl[31];
    S2_hi[59] = S1_hxl[31];
    S1_ca[31] = ((S1_ca[30] & S1_lo[31]) | (S1_lo[31] & S1_hi[31]) | (S1_ca[30] & S1_hi[31]));
    S1_cb[48] = ((S1_cb[47] & S1_is[31]) | (S1_is[31] & S1_lo[48]) | (S1_cb[47] & S1_lo[48]));
    S1_is[32] = S1_lo[49] ^ S1_out[49] ^ S1_cb[48];
    S1_hxl[32] = S1_is[32] ^ S1_ca[31];
    S1_hi[32] = S1_lo[32] ^ S1_hxl[32];
    S2_lo[32] = S1_lo[47] ^ S1_lo[11] ^ S1_hi[11] ^ S1_hxl[32];
    S2_hi[60] = S1_hxl[32];
    S2_hxl[32] = S1_lo[47] ^ S1_lo[11] ^ S1_hi[11] ^ S1_hxl[32] ^ S1_hxl[4];
    S1_ca[32] = ((S1_ca[31] & S1_lo[32]) | (S1_lo[32] & S1_hi[32]) | (S1_ca[31] & S1_hi[32]));
    S1_cb[49] = ((S1_cb[48] & S1_is[32]) | (S1_is[32] & S1_lo[49]) | (S1_cb[48] & S1_lo[49]));
    S1_is[33] = S1_lo[50] ^ S1_out[50] ^ S1_cb[49];
    S1_hxl[33] = S1_is[33] ^ S1_ca[32];
    S1_hi[33] = S1_lo[33] ^ S1_hxl[33];
    S2_lo[33] = S1_lo[48] ^ S1_lo[12] ^ S1_hi[12] ^ S1_hxl[33];
    S2_hi[61] = S1_hxl[33];
    S1_ca[33] = ((S1_ca[32] & S1_lo[33]) | (S1_lo[33] & S1_hi[33]) | (S1_ca[32] & S1_hi[33]));
    S1_cb[50] = ((S1_cb[49] & S1_is[33]) | (S1_is[33] & S1_lo[50]) | (S1_cb[49] & S1_lo[50]));
    S1_is[34] = S1_lo[51] ^ S1_out[51] ^ S1_cb[50];
    S1_hxl[34] = S1_is[34] ^ S1_ca[33];
    S1_hi[34] = S1_lo[34] ^ S1_hxl[34];
    S2_lo[34] = S1_lo[49] ^ S1_lo[13] ^ S1_hi[13] ^ S1_hxl[34];
    S2_hi[62] = S1_hxl[34];
    S1_ca[34] = ((S1_ca[33] & S1_lo[34]) | (S1_lo[34] & S1_hi[34]) | (S1_ca[33] & S1_hi[34]));
    S1_cb[51] = ((S1_cb[50] & S1_is[34]) | (S1_is[34] & S1_lo[51]) | (S1_cb[50] & S1_lo[51]));
    S2_ca[10] = S2_hxl[11] ^ S2_is[11];
    S2_ca[11] = ((S2_ca[10] & S2_lo[11]) | (S2_lo[11] & S2_hi[11]) | (S2_ca[10] & S2_hi[11]));
    S2_is[12] = S2_ca[11] ^ S2_hxl[12];
    S2_ca[12] = ((S2_ca[11] & S2_lo[12]) | (S2_lo[12] & S2_hi[12]) | (S2_ca[11] & S2_hi[12]));
    S2_is[13] = S2_ca[12] ^ S2_hxl[13];
    S2_ca[13] = ((S2_ca[12] & S2_lo[13]) | (S2_lo[13] & S2_hi[13]) | (S2_ca[12] & S2_hi[13]));
    S2_ca[31] = S2_hxl[32] ^ S2_is[32];
    S2_ca[32] = ((S2_ca[31] & S2_lo[32]) | (S2_lo[32] & S2_hi[32]) | (S2_ca[31] & S2_hi[32]));
    S2_cb[27] = S2_is[11] ^ S2_lo[28] ^ S2_out[28];
    S2_cb[28] = ((S2_cb[27] & S2_is[11]) | (S2_is[11] & S2_lo[28]) | (S2_cb[27] & S2_lo[28]));
    S2_lo[29] = S2_is[12] ^ S2_out[29] ^ S2_cb[28];
    S1_hxl[8] = S1_lo[44] ^ S1_lo[29] ^ S1_hi[29] ^ S2_lo[29];
    S2_hi[36] = S1_lo[44] ^ S1_lo[29] ^ S1_hi[29] ^ S2_lo[29];
    S2_cb[29] = ((S2_cb[28] & S2_is[12]) | (S2_is[12] & S2_lo[29]) | (S2_cb[28] & S2_lo[29]));
    S2_lo[30] = S2_is[13] ^ S2_out[30] ^ S2_cb[29];
    S1_hi[9] = S1_lo[45] ^ S1_lo[30] ^ S1_lo[9] ^ S1_hi[30] ^ S2_lo[30];
    S1_hxl[9] = S1_lo[45] ^ S1_lo[30] ^ S1_hi[30] ^ S2_lo[30];
    S2_hi[37] = S1_lo[45] ^ S1_lo[30] ^ S1_hi[30] ^ S2_lo[30];
    S1_ca[8] = S1_hxl[9] ^ S1_is[9];
    S1_ca[9] = ((S1_ca[8] & S1_lo[9]) | (S1_lo[9] & S1_hi[9]) | (S1_ca[8] & S1_hi[9]));
    S1_hxl[10] = S1_is[10] ^ S1_ca[9];
    S2_lo[31] = S1_lo[46] ^ S1_lo[31] ^ S1_hi[31] ^ S1_hxl[10];
    S2_hi[38] = S1_hxl[10];
    S2_cb[30] = ((S2_cb[29] & S2_is[13]) | (S2_is[13] & S2_lo[30]) | (S2_cb[29] & S2_lo[30]));
    S2_is[14] = S2_lo[31] ^ S2_out[31] ^ S2_cb[30];
    S2_hxl[14] = S2_is[14] ^ S2_ca[13];
    S1_hi[50] = S1_lo[50] ^ S1_lo[29] ^ S1_lo[14] ^ S1_hi[14] ^ S2_hxl[14];
    S1_hxl[50] = S1_lo[29] ^ S1_lo[14] ^ S1_hi[14] ^ S2_hxl[14];
    S2_lo[50] = S1_lo[14] ^ S1_lo[1] ^ S1_hi[29] ^ S1_hi[14] ^ S2_hxl[14];
    S2_hi[14] = S1_lo[29] ^ S1_lo[14] ^ S1_hi[14] ^ S2_hxl[14];
    S1_is[50] = S1_ca[49] ^ S1_hxl[50];
    S1_ca[50] = ((S1_ca[49] & S1_lo[50]) | (S1_lo[50] & S1_hi[50]) | (S1_ca[49] & S1_hi[50]));
    S1_lo[3] = S1_is[50] ^ S1_out[3] ^ S1_cb[2];
    S1_cb[3] = ((S1_cb[2] & S1_is[50]) | (S1_is[50] & S1_lo[3]) | (S1_cb[2] & S1_lo[3]));
    S2_ca[14] = ((S2_ca[13] & S2_lo[14]) | (S2_lo[14] & S2_hi[14]) | (S2_ca[13] & S2_hi[14]));
    S2_cb[31] = ((S2_cb[30] & S2_is[14]) | (S2_is[14] & S2_lo[31]) | (S2_cb[30] & S2_lo[31]));
    S2_is[15] = S2_lo[32] ^ S2_out[32] ^ S2_cb[31];
    S2_hxl[15] = S2_is[15] ^ S2_ca[14];
    S1_hi[51] = S1_lo[51] ^ S1_lo[30] ^ S1_lo[15] ^ S1_hi[15] ^ S2_hxl[15];
    S1_hxl[51] = S1_lo[30] ^ S1_lo[15] ^ S1_hi[15] ^ S2_hxl[15];
    S2_lo[51] = S1_lo[15] ^ S1_lo[2] ^ S1_hi[30] ^ S1_hi[15] ^ S2_hxl[15];
    S2_hi[15] = S1_lo[30] ^ S1_lo[15] ^ S1_hi[15] ^ S2_hxl[15];
    S1_is[51] = S1_ca[50] ^ S1_hxl[51];
    S1_ca[51] = ((S1_ca[50] & S1_lo[51]) | (S1_lo[51] & S1_hi[51]) | (S1_ca[50] & S1_hi[51]));
    S1_lo[4] = S1_is[51] ^ S1_out[4] ^ S1_cb[3];
    S1_hi[4] = S1_lo[4] ^ S1_hxl[4];
    S1_ca[4] = ((S1_ca[3] & S1_lo[4]) | (S1_lo[4] & S1_hi[4]) | (S1_ca[3] & S1_hi[4]));
    S1_cb[4] = ((S1_cb[3] & S1_is[51]) | (S1_is[51] & S1_lo[4]) | (S1_cb[3] & S1_lo[4]));
    S2_ca[15] = ((S2_ca[14] & S2_lo[15]) | (S2_lo[15] & S2_hi[15]) | (S2_ca[14] & S2_hi[15]));
    S2_cb[32] = ((S2_cb[31] & S2_is[15]) | (S2_is[15] & S2_lo[32]) | (S2_cb[31] & S2_lo[32]));
    S2_is[16] = S2_lo[33] ^ S2_out[33] ^ S2_cb[32];
    S2_hxl[16] = S2_is[16] ^ S2_ca[15];
    S1_hxl[52] = S1_lo[31] ^ S1_hxl[16] ^ S2_hxl[16];
    S2_lo[52] = S1_lo[3] ^ S1_hi[31] ^ S1_hxl[16] ^ S2_hxl[16];
    S2_hi[16] = S1_lo[31] ^ S1_hxl[16] ^ S2_hxl[16];
    S1_is[52] = S1_ca[51] ^ S1_hxl[52];
    S1_lo[5] = S1_is[52] ^ S1_out[5] ^ S1_cb[4];
    S1_cb[5] = ((S1_cb[4] & S1_is[52]) | (S1_is[52] & S1_lo[5]) | (S1_cb[4] & S1_lo[5]));
    S1_is[53] = S1_lo[6] ^ S1_out[6] ^ S1_cb[5];
    S1_cb[6] = ((S1_cb[5] & S1_is[53]) | (S1_is[53] & S1_lo[6]) | (S1_cb[5] & S1_lo[6]));
    S2_ca[16] = ((S2_ca[15] & S2_lo[16]) | (S2_lo[16] & S2_hi[16]) | (S2_ca[15] & S2_hi[16]));
    S2_cb[33] = ((S2_cb[32] & S2_is[16]) | (S2_is[16] & S2_lo[33]) | (S2_cb[32] & S2_lo[33]));
    S2_is[17] = S2_lo[34] ^ S2_out[34] ^ S2_cb[33];
    S2_hxl[17] = S2_is[17] ^ S2_ca[16];
    S2_cb[34] = ((S2_cb[33] & S2_is[17]) | (S2_is[17] & S2_lo[34]) | (S2_cb[33] & S2_lo[34]));
    S2_cb[48] = S2_is[32] ^ S2_lo[49] ^ S2_out[49];
    S2_cb[49] = ((S2_cb[48] & S2_is[32]) | (S2_is[32] & S2_lo[49]) | (S2_cb[48] & S2_lo[49]));
    S2_is[33] = S2_lo[50] ^ S2_out[50] ^ S2_cb[49];
    S2_hxl[33] = S2_is[33] ^ S2_ca[32];
    S1_hi[5] = S1_lo[48] ^ S1_lo[33] ^ S1_lo[12] ^ S1_lo[5] ^ S1_hi[33] ^ S1_hi[12] ^ S2_hxl[33];
    S1_hxl[5] = S1_lo[48] ^ S1_lo[33] ^ S1_lo[12] ^ S1_hi[33] ^ S1_hi[12] ^ S2_hxl[33];
    S2_hi[33] = S1_lo[48] ^ S1_lo[33] ^ S1_lo[12] ^ S1_hi[33] ^ S1_hi[12] ^ S2_hxl[33];
    S1_is[5] = S1_ca[4] ^ S1_hxl[5];
    S1_ca[5] = ((S1_ca[4] & S1_lo[5]) | (S1_lo[5] & S1_hi[5]) | (S1_ca[4] & S1_hi[5]));
    S1_lo[22] = S1_is[5] ^ S1_out[22] ^ S1_cb[21];
    S2_lo[7] = S1_lo[22] ^ S1_hxl[7];
    S1_cb[22] = ((S1_cb[21] & S1_is[5]) | (S1_is[5] & S1_lo[22]) | (S1_cb[21] & S1_lo[22]));
    S2_ca[33] = ((S2_ca[32] & S2_lo[33]) | (S2_lo[33] & S2_hi[33]) | (S2_ca[32] & S2_hi[33]));
    S2_cb[50] = ((S2_cb[49] & S2_is[33]) | (S2_is[33] & S2_lo[50]) | (S2_cb[49] & S2_lo[50]));
    S2_is[34] = S2_lo[51] ^ S2_out[51] ^ S2_cb[50];
    S2_hxl[34] = S2_is[34] ^ S2_ca[33];
    S1_hi[6] = S1_lo[49] ^ S1_lo[34] ^ S1_lo[13] ^ S1_lo[6] ^ S1_hi[34] ^ S1_hi[13] ^ S2_hxl[34];
    S1_hxl[6] = S1_lo[49] ^ S1_lo[34] ^ S1_lo[13] ^ S1_hi[34] ^ S1_hi[13] ^ S2_hxl[34];
    S2_lo[6] = S1_lo[49] ^ S1_lo[34] ^ S1_lo[21] ^ S1_lo[13] ^ S1_hi[34] ^ S1_hi[13] ^ S2_hxl[34];
    S2_hi[34] = S1_lo[49] ^ S1_lo[34] ^ S1_lo[13] ^ S1_hi[34] ^ S1_hi[13] ^ S2_hxl[34];
    S1_is[6] = S1_ca[5] ^ S1_hxl[6];
    S1_ca[6] = ((S1_ca[5] & S1_lo[6]) | (S1_lo[6] & S1_hi[6]) | (S1_ca[5] & S1_hi[6]));
    S1_is[7] = S1_ca[6] ^ S1_hxl[7];
    S1_lo[23] = S1_is[6] ^ S1_out[23] ^ S1_cb[22];
    S2_lo[8] = S1_lo[23] ^ S1_hxl[8];
    S1_cb[23] = ((S1_cb[22] & S1_is[6]) | (S1_is[6] & S1_lo[23]) | (S1_cb[22] & S1_lo[23]));
    S1_lo[24] = S1_is[7] ^ S1_out[24] ^ S1_cb[23];
    S2_lo[9] = S1_lo[24] ^ S1_lo[9] ^ S1_hi[9];
    S1_cb[24] = ((S1_cb[23] & S1_is[7]) | (S1_is[7] & S1_lo[24]) | (S1_cb[23] & S1_lo[24]));
    S2_ca[34] = ((S2_ca[33] & S2_lo[34]) | (S2_lo[34] & S2_hi[34]) | (S2_ca[33] & S2_hi[34]));
    S2_cb[5] = S2_is[53] ^ S2_lo[6] ^ S2_out[6];
    S2_cb[6] = ((S2_cb[5] & S2_is[53]) | (S2_is[53] & S2_lo[6]) | (S2_cb[5] & S2_lo[6]));
    S2_is[54] = S2_lo[7] ^ S2_out[7] ^ S2_cb[6];
    S2_cb[7] = ((S2_cb[6] & S2_is[54]) | (S2_is[54] & S2_lo[7]) | (S2_cb[6] & S2_lo[7]));
    S2_is[55] = S2_lo[8] ^ S2_out[8] ^ S2_cb[7];
    S2_cb[8] = ((S2_cb[7] & S2_is[55]) | (S2_is[55] & S2_lo[8]) | (S2_cb[7] & S2_lo[8]));
    S2_is[56] = S2_lo[9] ^ S2_out[9] ^ S2_cb[8];
    S2_cb[9] = ((S2_cb[8] & S2_is[56]) | (S2_is[56] & S2_lo[9]) | (S2_cb[8] & S2_lo[9]));
    S2_cb[51] = ((S2_cb[50] & S2_is[34]) | (S2_is[34] & S2_lo[51]) | (S2_cb[50] & S2_lo[51]));
    S2_is[35] = S2_lo[52] ^ S2_out[52] ^ S2_cb[51];
    S2_hxl[35] = S2_is[35] ^ S2_ca[34];
    S1_hi[35] = S1_lo[50] ^ S1_lo[35] ^ S1_lo[14] ^ S1_hi[14] ^ S1_hxl[7] ^ S2_hxl[35];
    S1_hxl[35] = S1_lo[50] ^ S1_lo[14] ^ S1_hi[14] ^ S1_hxl[7] ^ S2_hxl[35];
    S2_lo[35] = S1_hxl[7] ^ S2_hxl[35];
    S2_hi[63] = S1_lo[50] ^ S1_lo[14] ^ S1_hi[14] ^ S1_hxl[7] ^ S2_hxl[35];
    S1_is[35] = S1_ca[34] ^ S1_hxl[35];
    S1_ca[35] = ((S1_ca[34] & S1_lo[35]) | (S1_lo[35] & S1_hi[35]) | (S1_ca[34] & S1_hi[35]));
    S1_lo[52] = S1_is[35] ^ S1_out[52] ^ S1_cb[51];
    S1_hi[52] = S1_lo[52] ^ S1_hxl[52];
    S1_ca[52] = ((S1_ca[51] & S1_lo[52]) | (S1_lo[52] & S1_hi[52]) | (S1_ca[51] & S1_hi[52]));
    S1_hxl[53] = S1_is[53] ^ S1_ca[52];
    S1_hi[17] = S1_lo[32] ^ S1_lo[17] ^ S1_hxl[53] ^ S2_hxl[17];
    S1_hxl[17] = S1_lo[32] ^ S1_hxl[53] ^ S2_hxl[17];
    S2_lo[53] = S1_lo[32] ^ S1_lo[4] ^ S1_hi[32] ^ S1_hxl[53];
    S2_lo[17] = S1_hxl[53] ^ S2_hxl[17];
    S2_hi[45] = S1_lo[32] ^ S1_hxl[53] ^ S2_hxl[17];
    S2_hi[17] = S1_hxl[53];
    S1_ca[16] = S1_hxl[17] ^ S1_is[17];
    S1_ca[17] = ((S1_ca[16] & S1_lo[17]) | (S1_lo[17] & S1_hi[17]) | (S1_ca[16] & S1_hi[17]));
    S1_hxl[18] = S1_is[18] ^ S1_ca[17];
    S2_lo[18] = S1_lo[33] ^ S1_hxl[18];
    S2_hi[46] = S1_hxl[18];
    S1_cb[52] = ((S1_cb[51] & S1_is[35]) | (S1_is[35] & S1_lo[52]) | (S1_cb[51] & S1_lo[52]));
    S2_ca[17] = ((S2_ca[16] & S2_lo[17]) | (S2_lo[17] & S2_hi[17]) | (S2_ca[16] & S2_hi[17]));
    S2_ca[35] = ((S2_ca[34] & S2_lo[35]) | (S2_lo[35] & S2_hi[35]) | (S2_ca[34] & S2_hi[35]));
    S2_is[18] = S2_lo[35] ^ S2_out[35] ^ S2_cb[34];
    S2_hxl[18] = S2_is[18] ^ S2_ca[17];
    S1_hxl[54] = S1_lo[33] ^ S1_hxl[18] ^ S2_hxl[18];
    S2_lo[54] = S1_lo[5] ^ S1_hi[33] ^ S1_hxl[18] ^ S2_hxl[18];
    S2_hi[18] = S1_lo[33] ^ S1_hxl[18] ^ S2_hxl[18];
    S2_hxl[54] = S1_lo[26] ^ S1_lo[5] ^ S1_hi[33] ^ S1_hi[26] ^ S1_hxl[18] ^ S2_hxl[18];
    S2_ca[18] = ((S2_ca[17] & S2_lo[18]) | (S2_lo[18] & S2_hi[18]) | (S2_ca[17] & S2_hi[18]));
    S2_ca[53] = S2_hxl[54] ^ S2_is[54];
    S2_ca[54] = ((S2_ca[53] & S2_lo[54]) | (S2_lo[54] & S2_hi[54]) | (S2_ca[53] & S2_hi[54]));
    S2_hxl[55] = S2_is[55] ^ S2_ca[54];
    S1_hxl[55] = S1_lo[27] ^ S1_lo[6] ^ S1_hi[27] ^ S1_hxl[34] ^ S2_hxl[55];
    S2_lo[55] = S1_lo[27] ^ S1_hi[27] ^ S2_hxl[55];
    S2_hi[19] = S1_lo[27] ^ S1_lo[6] ^ S1_hi[27] ^ S1_hxl[34] ^ S2_hxl[55];
    S2_ca[55] = ((S2_ca[54] & S2_lo[55]) | (S2_lo[55] & S2_hi[55]) | (S2_ca[54] & S2_hi[55]));
    S2_hxl[56] = S2_is[56] ^ S2_ca[55];
    S2_lo[56] = S1_lo[28] ^ S1_hi[28] ^ S2_hxl[56];
    S2_ca[56] = ((S2_ca[55] & S2_lo[56]) | (S2_lo[56] & S2_hi[56]) | (S2_ca[55] & S2_hi[56]));
    S2_cb[35] = ((S2_cb[34] & S2_is[18]) | (S2_is[18] & S2_lo[35]) | (S2_cb[34] & S2_lo[35]));
    S2_cb[52] = ((S2_cb[51] & S2_is[35]) | (S2_is[35] & S2_lo[52]) | (S2_cb[51] & S2_lo[52]));
    S2_is[36] = S2_lo[53] ^ S2_out[53] ^ S2_cb[52];
    S2_hxl[36] = S2_is[36] ^ S2_ca[35];
    S1_hxl[36] = S1_lo[51] ^ S1_lo[15] ^ S1_hi[15] ^ S1_hxl[8] ^ S2_hxl[36];
    S2_lo[36] = S1_hxl[8] ^ S2_hxl[36];
    S2_hi[0] = S1_lo[51] ^ S1_lo[15] ^ S1_hi[15] ^ S1_hxl[8] ^ S2_hxl[36];
    S1_is[36] = S1_ca[35] ^ S1_hxl[36];
    S1_lo[53] = S1_is[36] ^ S1_out[53] ^ S1_cb[52];
    S1_hi[53] = S1_lo[53] ^ S1_hxl[53];
    S1_ca[53] = ((S1_ca[52] & S1_lo[53]) | (S1_lo[53] & S1_hi[53]) | (S1_ca[52] & S1_hi[53]));
    S1_is[54] = S1_ca[53] ^ S1_hxl[54];
    S1_lo[7] = S1_is[54] ^ S1_out[7] ^ S1_cb[6];
    S1_hi[7] = S1_lo[7] ^ S1_hxl[7];
    S1_hxl[56] = S1_lo[7] ^ S1_hxl[35] ^ S2_lo[56];
    S2_hi[20] = S1_lo[7] ^ S1_hxl[35] ^ S2_lo[56];
    S1_ca[7] = ((S1_ca[6] & S1_lo[7]) | (S1_lo[7] & S1_hi[7]) | (S1_ca[6] & S1_hi[7]));
    S1_is[8] = S1_ca[7] ^ S1_hxl[8];
    S1_cb[7] = ((S1_cb[6] & S1_is[54]) | (S1_is[54] & S1_lo[7]) | (S1_cb[6] & S1_lo[7]));
    S1_lo[25] = S1_is[8] ^ S1_out[25] ^ S1_cb[24];
    S2_lo[10] = S1_lo[25] ^ S1_hxl[10];
    S1_cb[53] = ((S1_cb[52] & S1_is[36]) | (S1_is[36] & S1_lo[53]) | (S1_cb[52] & S1_lo[53]));
    S2_ca[36] = ((S2_ca[35] & S2_lo[36]) | (S2_lo[36] & S2_hi[36]) | (S2_ca[35] & S2_hi[36]));
    S2_is[57] = S2_lo[10] ^ S2_out[10] ^ S2_cb[9];
    S2_hxl[57] = S2_is[57] ^ S2_ca[56];
    S2_lo[57] = S1_lo[29] ^ S1_hi[29] ^ S2_hxl[57];
    S2_ca[57] = ((S2_ca[56] & S2_lo[57]) | (S2_lo[57] & S2_hi[57]) | (S2_ca[56] & S2_hi[57]));
    S2_cb[10] = ((S2_cb[9] & S2_is[57]) | (S2_is[57] & S2_lo[10]) | (S2_cb[9] & S2_lo[10]));
    S2_is[58] = S2_lo[11] ^ S2_out[11] ^ S2_cb[10];
    S2_hxl[58] = S2_is[58] ^ S2_ca[57];
    S2_lo[58] = S1_lo[30] ^ S1_hi[30] ^ S2_hxl[58];
    S2_ca[58] = ((S2_ca[57] & S2_lo[58]) | (S2_lo[58] & S2_hi[58]) | (S2_ca[57] & S2_hi[58]));
    S2_cb[11] = ((S2_cb[10] & S2_is[58]) | (S2_is[58] & S2_lo[11]) | (S2_cb[10] & S2_lo[11]));
    S2_is[59] = S2_lo[12] ^ S2_out[12] ^ S2_cb[11];
    S2_hxl[59] = S2_is[59] ^ S2_ca[58];
    S2_lo[59] = S1_hxl[31] ^ S2_hxl[59];
    S2_ca[59] = ((S2_ca[58] & S2_lo[59]) | (S2_lo[59] & S2_hi[59]) | (S2_ca[58] & S2_hi[59]));
    S2_cb[12] = ((S2_cb[11] & S2_is[59]) | (S2_is[59] & S2_lo[12]) | (S2_cb[11] & S2_lo[12]));
    S2_is[60] = S2_lo[13] ^ S2_out[13] ^ S2_cb[12];
    S2_hxl[60] = S2_is[60] ^ S2_ca[59];
    S2_lo[60] = S1_lo[32] ^ S1_hi[32] ^ S2_hxl[60];
    S2_ca[60] = ((S2_ca[59] & S2_lo[60]) | (S2_lo[60] & S2_hi[60]) | (S2_ca[59] & S2_hi[60]));
    S2_cb[13] = ((S2_cb[12] & S2_is[60]) | (S2_is[60] & S2_lo[13]) | (S2_cb[12] & S2_lo[13]));
    S2_is[61] = S2_lo[14] ^ S2_out[14] ^ S2_cb[13];
    S2_hxl[61] = S2_is[61] ^ S2_ca[60];
    S2_lo[61] = S1_hxl[33] ^ S2_hxl[61];
    S2_ca[61] = ((S2_ca[60] & S2_lo[61]) | (S2_lo[61] & S2_hi[61]) | (S2_ca[60] & S2_hi[61]));
    S2_cb[14] = ((S2_cb[13] & S2_is[61]) | (S2_is[61] & S2_lo[14]) | (S2_cb[13] & S2_lo[14]));
    S2_is[62] = S2_lo[15] ^ S2_out[15] ^ S2_cb[14];
    S2_hxl[62] = S2_is[62] ^ S2_ca[61];
    S2_lo[62] = S1_lo[34] ^ S1_hi[34] ^ S2_hxl[62];
    S2_ca[62] = ((S2_ca[61] & S2_lo[62]) | (S2_lo[62] & S2_hi[62]) | (S2_ca[61] & S2_hi[62]));
    S2_cb[15] = ((S2_cb[14] & S2_is[62]) | (S2_is[62] & S2_lo[15]) | (S2_cb[14] & S2_lo[15]));
    S2_is[63] = S2_lo[16] ^ S2_out[16] ^ S2_cb[15];
    S2_hxl[63] = S2_is[63] ^ S2_ca[62];
    S2_lo[63] = S1_lo[35] ^ S1_hi[35] ^ S2_hxl[63];
    S2_ca[63] = ((S2_ca[62] & S2_lo[63]) | (S2_lo[63] & S2_hi[63]) | (S2_ca[62] & S2_hi[63]));
    S2_cb[16] = ((S2_cb[15] & S2_is[63]) | (S2_is[63] & S2_lo[16]) | (S2_cb[15] & S2_lo[16]));
    S2_is[0] = S2_lo[17] ^ S2_out[17] ^ S2_cb[16];
    S2_hxl[0] = S2_is[0];
    S1_hi[0] = S1_lo[15] ^ S1_lo[0] ^ S1_hxl[36] ^ S2_hxl[0];
    S1_hxl[0] = S1_lo[15] ^ S1_hxl[36] ^ S2_hxl[0];
    S2_lo[0] = S1_hxl[36] ^ S2_hxl[0];
    S2_hi[28] = S1_lo[15] ^ S1_hxl[36] ^ S2_hxl[0];
    S2_hxl[28] = S1_lo[43] ^ S1_lo[28] ^ S1_lo[15] ^ S1_lo[7] ^ S1_hi[28] ^ S1_hi[7] ^ S1_hxl[36] ^ S2_hxl[0];
    S1_is[0] = S1_hxl[0];
    S1_ca[0] = S1_lo[0] & S1_hi[0];
    S1_cb[16] = S1_is[0] ^ S1_lo[17] ^ S1_out[17];
    S1_cb[17] = ((S1_cb[16] & S1_is[0]) | (S1_is[0] & S1_lo[17]) | (S1_cb[16] & S1_lo[17]));
    S2_ca[0] = S2_lo[0] & S2_hi[0];
    S2_is[47] = S2_lo[0] ^ S2_out[0];
    S2_cb[0] = S2_is[47] & S2_lo[0];
    S2_cb[17] = ((S2_cb[16] & S2_is[0]) | (S2_is[0] & S2_lo[17]) | (S2_cb[16] & S2_lo[17]));
    S2_is[1] = S2_lo[18] ^ S2_out[18] ^ S2_cb[17];
    S2_hxl[1] = S2_is[1] ^ S2_ca[0];
    S2_cb[18] = ((S2_cb[17] & S2_is[1]) | (S2_is[1] & S2_lo[18]) | (S2_cb[17] & S2_lo[18]));
    S2_is[19] = S2_lo[36] ^ S2_out[36] ^ S2_cb[35];
    S2_hxl[19] = S2_is[19] ^ S2_ca[18];
    S1_hxl[19] = S1_lo[34] ^ S1_hxl[55] ^ S2_hxl[19];
    S2_lo[19] = S1_hxl[55] ^ S2_hxl[19];
    S2_hi[47] = S1_lo[34] ^ S1_hxl[55] ^ S2_hxl[19];
    S2_ca[19] = ((S2_ca[18] & S2_lo[19]) | (S2_lo[19] & S2_hi[19]) | (S2_ca[18] & S2_hi[19]));
    S2_is[2] = S2_lo[19] ^ S2_out[19] ^ S2_cb[18];
    S2_cb[19] = ((S2_cb[18] & S2_is[2]) | (S2_is[2] & S2_lo[19]) | (S2_cb[18] & S2_lo[19]));
    S2_cb[36] = ((S2_cb[35] & S2_is[19]) | (S2_is[19] & S2_lo[36]) | (S2_cb[35] & S2_lo[36]));
    S2_cb[53] = ((S2_cb[52] & S2_is[36]) | (S2_is[36] & S2_lo[53]) | (S2_cb[52] & S2_lo[53]));
    S2_is[37] = S2_lo[54] ^ S2_out[54] ^ S2_cb[53];
    S2_hxl[37] = S2_is[37] ^ S2_ca[36];
    S1_hxl[58] = S1_hi[52] ^ S1_hi[9] ^ S1_hxl[52] ^ S1_hxl[16] ^ S2_lo[58] ^ S2_hxl[37];
    S1_hxl[37] = S1_lo[9] ^ S1_hi[52] ^ S1_hi[9] ^ S1_hxl[52] ^ S1_hxl[16] ^ S2_hxl[37];
    S2_lo[37] = S1_lo[9] ^ S1_hi[9] ^ S2_hxl[37];
    S2_lo[1] = S1_lo[9] ^ S1_hi[52] ^ S1_hi[9] ^ S1_hxl[52] ^ S1_hxl[16] ^ S2_hxl[37] ^ S2_hxl[1];
    S2_hi[22] = S1_hi[52] ^ S1_hi[9] ^ S1_hxl[52] ^ S1_hxl[16] ^ S2_lo[58] ^ S2_hxl[37];
    S2_hi[1] = S1_lo[9] ^ S1_hi[52] ^ S1_hi[9] ^ S1_hxl[52] ^ S1_hxl[16] ^ S2_hxl[37];
    S2_ca[1] = ((S2_ca[0] & S2_lo[1]) | (S2_lo[1] & S2_hi[1]) | (S2_ca[0] & S2_hi[1]));
    S2_hxl[2] = S2_is[2] ^ S2_ca[1];
    S2_ca[37] = ((S2_ca[36] & S2_lo[37]) | (S2_lo[37] & S2_hi[37]) | (S2_ca[36] & S2_hi[37]));
    S2_is[48] = S2_lo[1] ^ S2_out[1] ^ S2_cb[0];
    S2_cb[1] = ((S2_cb[0] & S2_is[48]) | (S2_is[48] & S2_lo[1]) | (S2_cb[0] & S2_lo[1]));
    S2_is[20] = S2_lo[37] ^ S2_out[37] ^ S2_cb[36];
    S2_hxl[20] = S2_is[20] ^ S2_ca[19];
    S1_hxl[20] = S1_lo[35] ^ S1_hxl[56] ^ S2_hxl[20];
    S2_lo[20] = S1_hxl[56] ^ S2_hxl[20];
    S2_hi[48] = S1_lo[35] ^ S1_hxl[56] ^ S2_hxl[20];
    S2_ca[20] = ((S2_ca[19] & S2_lo[20]) | (S2_lo[20] & S2_hi[20]) | (S2_ca[19] & S2_hi[20]));
    S2_is[3] = S2_lo[20] ^ S2_out[20] ^ S2_cb[19];
    S2_cb[20] = ((S2_cb[19] & S2_is[3]) | (S2_is[3] & S2_lo[20]) | (S2_cb[19] & S2_lo[20]));
    S2_cb[37] = ((S2_cb[36] & S2_is[20]) | (S2_is[20] & S2_lo[37]) | (S2_cb[36] & S2_lo[37]));
    S2_cb[54] = ((S2_cb[53] & S2_is[37]) | (S2_is[37] & S2_lo[54]) | (S2_cb[53] & S2_lo[54]));
    S2_is[38] = S2_lo[55] ^ S2_out[55] ^ S2_cb[54];
    S2_hxl[38] = S2_is[38] ^ S2_ca[37];
    S1_hi[2] = S1_lo[2] ^ S1_hi[53] ^ S1_hi[17] ^ S1_hxl[53] ^ S1_hxl[10] ^ S2_hxl[38] ^ S2_hxl[2];
    S1_hxl[38] = S1_lo[17] ^ S1_hi[53] ^ S1_hi[17] ^ S1_hxl[53] ^ S1_hxl[10] ^ S2_hxl[38];
    S1_hxl[2] = S1_hi[53] ^ S1_hi[17] ^ S1_hxl[53] ^ S1_hxl[10] ^ S2_hxl[38] ^ S2_hxl[2];
    S2_lo[38] = S1_hxl[10] ^ S2_hxl[38];
    S2_lo[2] = S1_lo[17] ^ S1_hi[53] ^ S1_hi[17] ^ S1_hxl[53] ^ S1_hxl[10] ^ S2_hxl[38] ^ S2_hxl[2];
    S2_hi[30] = S1_hi[53] ^ S1_hi[17] ^ S1_hxl[53] ^ S1_hxl[10] ^ S2_hxl[38] ^ S2_hxl[2];
    S2_hi[2] = S1_lo[17] ^ S1_hi[53] ^ S1_hi[17] ^ S1_hxl[53] ^ S1_hxl[10] ^ S2_hxl[38];
    S2_hxl[30] = S1_lo[45] ^ S1_lo[9] ^ S1_hi[53] ^ S1_hi[17] ^ S1_hi[9] ^ S1_hxl[53] ^ S1_hxl[30] ^ S1_hxl[10] ^ S2_hxl[38] ^ S2_hxl[2];
    S2_ca[2] = ((S2_ca[1] & S2_lo[2]) | (S2_lo[2] & S2_hi[2]) | (S2_ca[1] & S2_hi[2]));
    S2_hxl[3] = S2_is[3] ^ S2_ca[2];
    S2_ca[38] = ((S2_ca[37] & S2_lo[38]) | (S2_lo[38] & S2_hi[38]) | (S2_ca[37] & S2_hi[38]));
    S2_is[49] = S2_lo[2] ^ S2_out[2] ^ S2_cb[1];
    S2_cb[2] = ((S2_cb[1] & S2_is[49]) | (S2_is[49] & S2_lo[2]) | (S2_cb[1] & S2_lo[2]));
    S2_is[21] = S2_lo[38] ^ S2_out[38] ^ S2_cb[37];
    S2_hxl[21] = S2_is[21] ^ S2_ca[20];
    S2_cb[38] = ((S2_cb[37] & S2_is[21]) | (S2_is[21] & S2_lo[38]) | (S2_cb[37] & S2_lo[38]));
    S2_cb[55] = ((S2_cb[54] & S2_is[38]) | (S2_is[38] & S2_lo[55]) | (S2_cb[54] & S2_lo[55]));
    S2_is[39] = S2_lo[56] ^ S2_out[56] ^ S2_cb[55];
    S2_hxl[39] = S2_is[39] ^ S2_ca[38];
    S2_lo[39] = S1_lo[11] ^ S1_hi[11] ^ S2_hxl[39];
    S2_ca[39] = ((S2_ca[38] & S2_lo[39]) | (S2_lo[39] & S2_hi[39]) | (S2_ca[38] & S2_hi[39]));
    S2_is[22] = S2_lo[39] ^ S2_out[39] ^ S2_cb[38];
    S2_cb[39] = ((S2_cb[38] & S2_is[22]) | (S2_is[22] & S2_lo[39]) | (S2_cb[38] & S2_lo[39]));
    S2_cb[56] = ((S2_cb[55] & S2_is[39]) | (S2_is[39] & S2_lo[56]) | (S2_cb[55] & S2_lo[56]));
    S2_is[40] = S2_lo[57] ^ S2_out[57] ^ S2_cb[56];
    S2_hxl[40] = S2_is[40] ^ S2_ca[39];
    S2_lo[40] = S1_hxl[12] ^ S2_hxl[40];
    S2_ca[40] = ((S2_ca[39] & S2_lo[40]) | (S2_lo[40] & S2_hi[40]) | (S2_ca[39] & S2_hi[40]));
    S2_is[23] = S2_lo[40] ^ S2_out[40] ^ S2_cb[39];
    S2_cb[40] = ((S2_cb[39] & S2_is[23]) | (S2_is[23] & S2_lo[40]) | (S2_cb[39] & S2_lo[40]));
    S2_cb[57] = ((S2_cb[56] & S2_is[40]) | (S2_is[40] & S2_lo[57]) | (S2_cb[56] & S2_lo[57]));
    S2_is[41] = S2_lo[58] ^ S2_out[58] ^ S2_cb[57];
    S2_hxl[41] = S2_is[41] ^ S2_ca[40];
    S2_lo[41] = S1_hxl[13] ^ S2_hxl[41];
    S2_ca[41] = ((S2_ca[40] & S2_lo[41]) | (S2_lo[41] & S2_hi[41]) | (S2_ca[40] & S2_hi[41]));
    S2_is[24] = S2_lo[41] ^ S2_out[41] ^ S2_cb[40];
    S2_cb[41] = ((S2_cb[40] & S2_is[24]) | (S2_is[24] & S2_lo[41]) | (S2_cb[40] & S2_lo[41]));
    S2_cb[58] = ((S2_cb[57] & S2_is[41]) | (S2_is[41] & S2_lo[58]) | (S2_cb[57] & S2_lo[58]));
    S2_is[42] = S2_lo[59] ^ S2_out[59] ^ S2_cb[58];
    S2_hxl[42] = S2_is[42] ^ S2_ca[41];
    S2_lo[42] = S1_lo[14] ^ S1_hi[14] ^ S2_hxl[42];
    S2_ca[42] = ((S2_ca[41] & S2_lo[42]) | (S2_lo[42] & S2_hi[42]) | (S2_ca[41] & S2_hi[42]));
    S2_is[25] = S2_lo[42] ^ S2_out[42] ^ S2_cb[41];
    S2_cb[42] = ((S2_cb[41] & S2_is[25]) | (S2_is[25] & S2_lo[42]) | (S2_cb[41] & S2_lo[42]));
    S2_cb[59] = ((S2_cb[58] & S2_is[42]) | (S2_is[42] & S2_lo[59]) | (S2_cb[58] & S2_lo[59]));
    S2_is[43] = S2_lo[60] ^ S2_out[60] ^ S2_cb[59];
    S2_hxl[43] = S2_is[43] ^ S2_ca[42];
    S2_lo[43] = S1_lo[15] ^ S1_hi[15] ^ S2_hxl[43];
    S2_ca[43] = ((S2_ca[42] & S2_lo[43]) | (S2_lo[43] & S2_hi[43]) | (S2_ca[42] & S2_hi[43]));
    S2_is[26] = S2_lo[43] ^ S2_out[43] ^ S2_cb[42];
    S2_cb[43] = ((S2_cb[42] & S2_is[26]) | (S2_is[26] & S2_lo[43]) | (S2_cb[42] & S2_lo[43]));
    S2_cb[60] = ((S2_cb[59] & S2_is[43]) | (S2_is[43] & S2_lo[60]) | (S2_cb[59] & S2_lo[60]));
    S2_is[44] = S2_lo[61] ^ S2_out[61] ^ S2_cb[60];
    S2_hxl[44] = S2_is[44] ^ S2_ca[43];
    S2_lo[44] = S1_hxl[16] ^ S2_hxl[44];
    S2_ca[44] = ((S2_ca[43] & S2_lo[44]) | (S2_lo[44] & S2_hi[44]) | (S2_ca[43] & S2_hi[44]));
    S2_is[27] = S2_lo[44] ^ S2_out[44] ^ S2_cb[43];
    S2_cb[44] = ((S2_cb[43] & S2_is[27]) | (S2_is[27] & S2_lo[44]) | (S2_cb[43] & S2_lo[44]));
    S2_cb[61] = ((S2_cb[60] & S2_is[44]) | (S2_is[44] & S2_lo[61]) | (S2_cb[60] & S2_lo[61]));
    S2_is[45] = S2_lo[62] ^ S2_out[62] ^ S2_cb[61];
    S2_hxl[45] = S2_is[45] ^ S2_ca[44];
    S2_lo[45] = S1_lo[17] ^ S1_hi[17] ^ S2_hxl[45];
    S2_ca[45] = ((S2_ca[44] & S2_lo[45]) | (S2_lo[45] & S2_hi[45]) | (S2_ca[44] & S2_hi[45]));
    S2_is[28] = S2_lo[45] ^ S2_out[45] ^ S2_cb[44];
    S2_ca[27] = S2_hxl[28] ^ S2_is[28];
    S2_ca[28] = ((S2_ca[27] & S2_lo[28]) | (S2_lo[28] & S2_hi[28]) | (S2_ca[27] & S2_hi[28]));
    S2_cb[45] = ((S2_cb[44] & S2_is[28]) | (S2_is[28] & S2_lo[45]) | (S2_cb[44] & S2_lo[45]));
    S2_cb[62] = ((S2_cb[61] & S2_is[45]) | (S2_is[45] & S2_lo[62]) | (S2_cb[61] & S2_lo[62]));
    S2_is[46] = S2_lo[63] ^ S2_out[63] ^ S2_cb[62];
    S2_hxl[46] = S2_is[46] ^ S2_ca[45];
    S2_lo[46] = S1_hxl[18] ^ S2_hxl[46];
    S2_ca[46] = ((S2_ca[45] & S2_lo[46]) | (S2_lo[46] & S2_hi[46]) | (S2_ca[45] & S2_hi[46]));
    S2_hxl[47] = S2_is[47] ^ S2_ca[46];
    S1_lo[62] = S1_hxl[47] ^ S1_hxl[26] ^ S1_hxl[19] ^ S2_hxl[47];
    S2_lo[47] = S1_hxl[19] ^ S2_hxl[47];
    S2_ca[47] = ((S2_ca[46] & S2_lo[47]) | (S2_lo[47] & S2_hi[47]) | (S2_ca[46] & S2_hi[47]));
    S2_hxl[48] = S2_is[48] ^ S2_ca[47];
    S1_lo[63] = S1_hxl[48] ^ S1_hxl[27] ^ S1_hxl[20] ^ S2_hxl[48];
    S2_lo[48] = S1_hxl[20] ^ S2_hxl[48];
    S2_ca[48] = ((S2_ca[47] & S2_lo[48]) | (S2_lo[48] & S2_hi[48]) | (S2_ca[47] & S2_hi[48]));
    S2_hxl[49] = S2_is[49] ^ S2_ca[48];
    S1_hi[21] = S1_lo[21] ^ S1_lo[0] ^ S1_hxl[49] ^ S1_hxl[28] ^ S2_hxl[49];
    S1_hxl[21] = S1_lo[0] ^ S1_hxl[49] ^ S1_hxl[28] ^ S2_hxl[49];
    S2_hi[49] = S1_lo[0] ^ S1_hxl[49] ^ S1_hxl[28] ^ S2_hxl[49];
    S2_ca[49] = ((S2_ca[48] & S2_lo[49]) | (S2_lo[49] & S2_hi[49]) | (S2_ca[48] & S2_hi[49]));
    S2_is[29] = S2_lo[46] ^ S2_out[46] ^ S2_cb[45];
    S2_hxl[29] = S2_is[29] ^ S2_ca[28];
    S1_lo[16] = S1_lo[44] ^ S1_lo[29] ^ S1_hi[29] ^ S1_hxl[8] ^ S2_lo[1] ^ S2_hxl[29];
    S1_hi[16] = S1_lo[44] ^ S1_lo[29] ^ S1_hi[29] ^ S1_hxl[16] ^ S1_hxl[8] ^ S2_lo[1] ^ S2_hxl[29];
    S1_hi[1] = S1_lo[44] ^ S1_lo[29] ^ S1_lo[1] ^ S1_hi[29] ^ S1_hxl[8] ^ S2_hxl[29];
    S1_hxl[1] = S1_lo[44] ^ S1_lo[29] ^ S1_hi[29] ^ S1_hxl[8] ^ S2_hxl[29];
    S2_hi[29] = S1_lo[44] ^ S1_lo[29] ^ S1_hi[29] ^ S1_hxl[8] ^ S2_hxl[29];
    S1_is[1] = S1_ca[0] ^ S1_hxl[1];
    S1_ca[1] = ((S1_ca[0] & S1_lo[1]) | (S1_lo[1] & S1_hi[1]) | (S1_ca[0] & S1_hi[1]));
    S1_is[2] = S1_ca[1] ^ S1_hxl[2];
    S1_ca[2] = ((S1_ca[1] & S1_lo[2]) | (S1_lo[2] & S1_hi[2]) | (S1_ca[1] & S1_hi[2]));
    S1_lo[18] = S1_is[1] ^ S1_out[18] ^ S1_cb[17];
    S1_hi[18] = S1_lo[18] ^ S1_hxl[18];
    S1_ca[18] = ((S1_ca[17] & S1_lo[18]) | (S1_lo[18] & S1_hi[18]) | (S1_ca[17] & S1_hi[18]));
    S1_is[19] = S1_ca[18] ^ S1_hxl[19];
    S1_cb[18] = ((S1_cb[17] & S1_is[1]) | (S1_is[1] & S1_lo[18]) | (S1_cb[17] & S1_lo[18]));
    S1_lo[19] = S1_is[2] ^ S1_out[19] ^ S1_cb[18];
    S1_hi[19] = S1_lo[19] ^ S1_hxl[19];
    S2_lo[4] = S1_lo[19] ^ S1_lo[4] ^ S1_hi[4];
    S1_ca[19] = ((S1_ca[18] & S1_lo[19]) | (S1_lo[19] & S1_hi[19]) | (S1_ca[18] & S1_hi[19]));
    S1_is[20] = S1_ca[19] ^ S1_hxl[20];
    S1_cb[19] = ((S1_cb[18] & S1_is[2]) | (S1_is[2] & S1_lo[19]) | (S1_cb[18] & S1_lo[19]));
    S1_lo[36] = S1_is[19] ^ S1_out[36] ^ S1_cb[35];
    S1_lo[8] = S1_lo[36] ^ S1_lo[0] ^ S1_hi[0] ^ S1_hxl[36] ^ S1_hxl[21] ^ S2_lo[57] ^ S2_hxl[21];
    S1_hi[36] = S1_lo[36] ^ S1_hxl[36];
    S1_hi[8] = S1_lo[36] ^ S1_lo[0] ^ S1_hi[0] ^ S1_hxl[36] ^ S1_hxl[21] ^ S1_hxl[8] ^ S2_lo[57] ^ S2_hxl[21];
    S1_hxl[57] = S1_lo[36] ^ S1_lo[0] ^ S1_hi[0] ^ S1_hxl[21] ^ S2_hxl[21];
    S2_lo[21] = S1_lo[36] ^ S1_lo[0] ^ S1_hi[0] ^ S1_hxl[21];
    S2_hi[21] = S1_lo[36] ^ S1_lo[0] ^ S1_hi[0] ^ S1_hxl[21] ^ S2_hxl[21];
    S1_ca[36] = ((S1_ca[35] & S1_lo[36]) | (S1_lo[36] & S1_hi[36]) | (S1_ca[35] & S1_hi[36]));
    S1_is[37] = S1_ca[36] ^ S1_hxl[37];
    S1_is[55] = S1_lo[8] ^ S1_out[8] ^ S1_cb[7];
    S1_ca[54] = S1_hxl[55] ^ S1_is[55];
    S1_cb[8] = ((S1_cb[7] & S1_is[55]) | (S1_is[55] & S1_lo[8]) | (S1_cb[7] & S1_lo[8]));
    S1_is[56] = S1_lo[9] ^ S1_out[9] ^ S1_cb[8];
    S1_ca[55] = S1_hxl[56] ^ S1_is[56];
    S1_cb[9] = ((S1_cb[8] & S1_is[56]) | (S1_is[56] & S1_lo[9]) | (S1_cb[8] & S1_lo[9]));
    S1_cb[36] = ((S1_cb[35] & S1_is[19]) | (S1_is[19] & S1_lo[36]) | (S1_cb[35] & S1_lo[36]));
    S1_lo[37] = S1_is[20] ^ S1_out[37] ^ S1_cb[36];
    S1_hi[37] = S1_lo[37] ^ S1_hxl[37];
    S1_ca[37] = ((S1_ca[36] & S1_lo[37]) | (S1_lo[37] & S1_hi[37]) | (S1_ca[36] & S1_hi[37]));
    S1_is[38] = S1_ca[37] ^ S1_hxl[38];
    S1_cb[37] = ((S1_cb[36] & S1_is[20]) | (S1_is[20] & S1_lo[37]) | (S1_cb[36] & S1_lo[37]));
    S1_lo[54] = S1_is[37] ^ S1_out[54] ^ S1_cb[53];
    S1_hi[54] = S1_lo[54] ^ S1_hxl[54];
    S1_hi[3] = S1_lo[54] ^ S1_lo[3] ^ S1_hi[18] ^ S2_lo[39] ^ S2_hxl[3];
    S1_hxl[60] = S1_lo[54] ^ S1_lo[18] ^ S1_lo[11] ^ S1_hi[18] ^ S2_lo[60] ^ S2_lo[39];
    S1_hxl[39] = S1_lo[54] ^ S1_lo[18] ^ S1_hi[18] ^ S2_lo[39];
    S1_hxl[3] = S1_lo[54] ^ S1_hi[18] ^ S2_lo[39] ^ S2_hxl[3];
    S2_lo[3] = S1_lo[54] ^ S1_lo[18] ^ S1_hi[18] ^ S2_lo[39] ^ S2_hxl[3];
    S2_hi[31] = S1_lo[54] ^ S1_hi[18] ^ S2_lo[39] ^ S2_hxl[3];
    S2_hi[24] = S1_lo[54] ^ S1_lo[18] ^ S1_lo[11] ^ S1_hi[18] ^ S2_lo[60] ^ S2_lo[39];
    S2_hi[3] = S1_lo[54] ^ S1_lo[18] ^ S1_hi[18] ^ S2_lo[39];
    S2_hxl[31] = S1_lo[54] ^ S1_lo[46] ^ S1_hi[18] ^ S1_hxl[31] ^ S1_hxl[10] ^ S2_lo[39] ^ S2_hxl[3];
    S1_is[3] = S1_ca[2] ^ S1_hxl[3];
    S1_lo[20] = S1_is[3] ^ S1_out[20] ^ S1_cb[19];
    S1_hi[20] = S1_lo[20] ^ S1_hxl[20];
    S2_lo[5] = S1_lo[20] ^ S1_hxl[5];
    S1_ca[20] = ((S1_ca[19] & S1_lo[20]) | (S1_lo[20] & S1_hi[20]) | (S1_ca[19] & S1_hi[20]));
    S1_is[21] = S1_ca[20] ^ S1_hxl[21];
    S1_ca[21] = ((S1_ca[20] & S1_lo[21]) | (S1_lo[21] & S1_hi[21]) | (S1_ca[20] & S1_hi[21]));
    S1_lo[38] = S1_is[21] ^ S1_out[38] ^ S1_cb[37];
    S1_hi[38] = S1_lo[38] ^ S1_hxl[38];
    S1_ca[38] = ((S1_ca[37] & S1_lo[38]) | (S1_lo[38] & S1_hi[38]) | (S1_ca[37] & S1_hi[38]));
    S1_is[39] = S1_ca[38] ^ S1_hxl[39];
    S1_cb[38] = ((S1_cb[37] & S1_is[21]) | (S1_is[21] & S1_lo[38]) | (S1_cb[37] & S1_lo[38]));
    S1_cb[54] = ((S1_cb[53] & S1_is[37]) | (S1_is[37] & S1_lo[54]) | (S1_cb[53] & S1_lo[54]));
    S1_lo[55] = S1_is[38] ^ S1_out[55] ^ S1_cb[54];
    S1_hi[55] = S1_lo[55] ^ S1_hxl[55];
    S1_hxl[61] = S1_lo[55] ^ S1_lo[19] ^ S1_lo[12] ^ S1_hi[19] ^ S2_lo[61] ^ S2_lo[40];
    S1_hxl[40] = S1_lo[55] ^ S1_lo[19] ^ S1_hi[19] ^ S2_lo[40];
    S2_hi[25] = S1_lo[55] ^ S1_lo[19] ^ S1_lo[12] ^ S1_hi[19] ^ S2_lo[61] ^ S2_lo[40];
    S2_hi[4] = S1_lo[55] ^ S1_lo[19] ^ S1_hi[19] ^ S2_lo[40];
    S2_hxl[4] = S1_lo[55] ^ S1_lo[4] ^ S1_hi[19] ^ S1_hi[4] ^ S2_lo[40];
    S1_cb[55] = ((S1_cb[54] & S1_is[38]) | (S1_is[38] & S1_lo[55]) | (S1_cb[54] & S1_lo[55]));
    S1_lo[56] = S1_is[39] ^ S1_out[56] ^ S1_cb[55];
    S1_hi[62] = S1_lo[62] ^ S1_lo[56] ^ S1_lo[20] ^ S1_lo[13] ^ S1_hi[20] ^ S2_lo[62] ^ S2_lo[41];
    S1_hi[56] = S1_lo[56] ^ S1_hxl[56];
    S1_hxl[62] = S1_lo[56] ^ S1_lo[20] ^ S1_lo[13] ^ S1_hi[20] ^ S2_lo[62] ^ S2_lo[41];
    S1_hxl[41] = S1_lo[56] ^ S1_lo[20] ^ S1_hi[20] ^ S2_lo[41];
    S2_hi[26] = S1_lo[56] ^ S1_lo[20] ^ S1_lo[13] ^ S1_hi[20] ^ S2_lo[62] ^ S2_lo[41];
    S2_hi[5] = S1_lo[56] ^ S1_lo[20] ^ S1_hi[20] ^ S2_lo[41];
    S2_hxl[5] = S1_lo[56] ^ S1_lo[5] ^ S1_hi[20] ^ S1_hi[5] ^ S2_lo[41];
    S1_ca[56] = ((S1_ca[55] & S1_lo[56]) | (S1_lo[56] & S1_hi[56]) | (S1_ca[55] & S1_hi[56]));
    S1_is[57] = S1_ca[56] ^ S1_hxl[57];
    S1_lo[10] = S1_is[57] ^ S1_out[10] ^ S1_cb[9];
    S1_hi[10] = S1_lo[10] ^ S1_hxl[10];
    S1_hxl[59] = S1_lo[10] ^ S1_hxl[38] ^ S2_lo[59];
    S2_hi[23] = S1_lo[10] ^ S1_hxl[38] ^ S2_lo[59];
    S1_cb[10] = ((S1_cb[9] & S1_is[57]) | (S1_is[57] & S1_lo[10]) | (S1_cb[9] & S1_lo[10]));
    S1_is[58] = S1_lo[11] ^ S1_out[11] ^ S1_cb[10];
    S1_ca[57] = S1_hxl[58] ^ S1_is[58];
    S1_cb[11] = ((S1_cb[10] & S1_is[58]) | (S1_is[58] & S1_lo[11]) | (S1_cb[10] & S1_lo[11]));
    S1_is[59] = S1_lo[12] ^ S1_out[12] ^ S1_cb[11];
    S1_ca[58] = S1_hxl[59] ^ S1_is[59];
    S1_cb[12] = ((S1_cb[11] & S1_is[59]) | (S1_is[59] & S1_lo[12]) | (S1_cb[11] & S1_lo[12]));
    S1_is[60] = S1_lo[13] ^ S1_out[13] ^ S1_cb[12];
    S1_ca[59] = S1_hxl[60] ^ S1_is[60];
    S1_cb[13] = ((S1_cb[12] & S1_is[60]) | (S1_is[60] & S1_lo[13]) | (S1_cb[12] & S1_lo[13]));
    S1_is[61] = S1_lo[14] ^ S1_out[14] ^ S1_cb[13];
    S1_ca[60] = S1_hxl[61] ^ S1_is[61];
    S1_cb[14] = ((S1_cb[13] & S1_is[61]) | (S1_is[61] & S1_lo[14]) | (S1_cb[13] & S1_lo[14]));
    S1_is[62] = S1_lo[15] ^ S1_out[15] ^ S1_cb[14];
    S1_ca[61] = S1_hxl[62] ^ S1_is[62];
    S1_ca[62] = ((S1_ca[61] & S1_lo[62]) | (S1_lo[62] & S1_hi[62]) | (S1_ca[61] & S1_hi[62]));
    S1_cb[15] = ((S1_cb[14] & S1_is[62]) | (S1_is[62] & S1_lo[15]) | (S1_cb[14] & S1_lo[15]));
    S1_is[63] = S1_lo[16] ^ S1_out[16] ^ S1_cb[15];
    S1_hxl[63] = S1_is[63] ^ S1_ca[62];
    S1_lo[57] = S1_lo[21] ^ S1_lo[14] ^ S1_hi[21] ^ S1_hxl[63] ^ S2_lo[63] ^ S2_lo[42];
    S1_hi[63] = S1_lo[63] ^ S1_hxl[63];
    S1_hi[57] = S1_lo[21] ^ S1_lo[14] ^ S1_hi[21] ^ S1_hxl[63] ^ S1_hxl[57] ^ S2_lo[63] ^ S2_lo[42];
    S1_hxl[42] = S1_lo[14] ^ S1_hxl[63] ^ S2_lo[63];
    S2_hi[27] = S1_hxl[63];
    S2_hi[6] = S1_lo[14] ^ S1_hxl[63] ^ S2_lo[63];
    S2_hxl[6] = S1_lo[21] ^ S1_lo[14] ^ S1_lo[6] ^ S1_hi[6] ^ S1_hxl[63] ^ S2_lo[63];
    S1_ca[63] = ((S1_ca[62] & S1_lo[63]) | (S1_lo[63] & S1_hi[63]) | (S1_ca[62] & S1_hi[63]));
    S1_cb[56] = ((S1_cb[55] & S1_is[39]) | (S1_is[39] & S1_lo[56]) | (S1_cb[55] & S1_lo[56]));
    S1_is[40] = S1_lo[57] ^ S1_out[57] ^ S1_cb[56];
    S1_ca[39] = S1_hxl[40] ^ S1_is[40];
    S1_cb[57] = ((S1_cb[56] & S1_is[40]) | (S1_is[40] & S1_lo[57]) | (S1_cb[56] & S1_lo[57]));
    S2_ca[3] = ((S2_ca[2] & S2_lo[3]) | (S2_lo[3] & S2_hi[3]) | (S2_ca[2] & S2_hi[3]));
    S2_is[4] = S2_ca[3] ^ S2_hxl[4];
    S2_ca[4] = ((S2_ca[3] & S2_lo[4]) | (S2_lo[4] & S2_hi[4]) | (S2_ca[3] & S2_hi[4]));
    S2_is[5] = S2_ca[4] ^ S2_hxl[5];
    S2_ca[5] = ((S2_ca[4] & S2_lo[5]) | (S2_lo[5] & S2_hi[5]) | (S2_ca[4] & S2_hi[5]));
    S2_is[6] = S2_ca[5] ^ S2_hxl[6];
    S2_ca[6] = ((S2_ca[5] & S2_lo[6]) | (S2_lo[6] & S2_hi[6]) | (S2_ca[5] & S2_hi[6]));
    S2_ca[21] = ((S2_ca[20] & S2_lo[21]) | (S2_lo[21] & S2_hi[21]) | (S2_ca[20] & S2_hi[21]));
    S2_hxl[22] = S2_is[22] ^ S2_ca[21];
    S1_hi[22] = S1_lo[37] ^ S1_lo[22] ^ S1_lo[1] ^ S1_hi[1] ^ S1_hxl[58] ^ S2_hxl[22];
    S1_hxl[22] = S1_lo[37] ^ S1_lo[1] ^ S1_hi[1] ^ S1_hxl[58] ^ S2_hxl[22];
    S2_lo[22] = S1_hxl[58] ^ S2_hxl[22];
    S2_hi[50] = S1_lo[37] ^ S1_lo[1] ^ S1_hi[1] ^ S1_hxl[58] ^ S2_hxl[22];
    S2_hxl[50] = S1_lo[50] ^ S1_lo[37] ^ S1_lo[29] ^ S1_hi[50] ^ S1_hi[29] ^ S1_hi[1] ^ S1_hxl[58] ^ S2_hxl[22];
    S1_is[22] = S1_ca[21] ^ S1_hxl[22];
    S1_ca[22] = ((S1_ca[21] & S1_lo[22]) | (S1_lo[22] & S1_hi[22]) | (S1_ca[21] & S1_hi[22]));
    S1_lo[39] = S1_is[22] ^ S1_out[39] ^ S1_cb[38];
    S1_hi[39] = S1_lo[39] ^ S1_hxl[39];
    S1_cb[39] = ((S1_cb[38] & S1_is[22]) | (S1_is[22] & S1_lo[39]) | (S1_cb[38] & S1_lo[39]));
    S2_ca[22] = ((S2_ca[21] & S2_lo[22]) | (S2_lo[22] & S2_hi[22]) | (S2_ca[21] & S2_hi[22]));
    S2_hxl[23] = S2_is[23] ^ S2_ca[22];
    S1_hi[23] = S1_lo[38] ^ S1_lo[23] ^ S1_lo[2] ^ S1_hi[2] ^ S1_hxl[59] ^ S2_hxl[23];
    S1_hxl[23] = S1_lo[38] ^ S1_lo[2] ^ S1_hi[2] ^ S1_hxl[59] ^ S2_hxl[23];
    S2_lo[23] = S1_hxl[59] ^ S2_hxl[23];
    S2_hi[51] = S1_lo[38] ^ S1_lo[2] ^ S1_hi[2] ^ S1_hxl[59] ^ S2_hxl[23];
    S2_hxl[51] = S1_lo[51] ^ S1_lo[38] ^ S1_lo[30] ^ S1_hi[51] ^ S1_hi[30] ^ S1_hi[2] ^ S1_hxl[59] ^ S2_hxl[23];
    S1_is[23] = S1_ca[22] ^ S1_hxl[23];
    S1_ca[23] = ((S1_ca[22] & S1_lo[23]) | (S1_lo[23] & S1_hi[23]) | (S1_ca[22] & S1_hi[23]));
    S1_lo[40] = S1_is[23] ^ S1_out[40] ^ S1_cb[39];
    S1_hi[40] = S1_lo[40] ^ S1_hxl[40];
    S1_ca[40] = ((S1_ca[39] & S1_lo[40]) | (S1_lo[40] & S1_hi[40]) | (S1_ca[39] & S1_hi[40]));
    S1_is[41] = S1_ca[40] ^ S1_hxl[41];
    S1_cb[40] = ((S1_cb[39] & S1_is[23]) | (S1_is[23] & S1_lo[40]) | (S1_cb[39] & S1_lo[40]));
    S1_lo[58] = S1_is[41] ^ S1_out[58] ^ S1_cb[57];
    S1_hi[58] = S1_lo[58] ^ S1_hxl[58];
    S1_hi[43] = S1_lo[58] ^ S1_lo[43] ^ S1_lo[22] ^ S1_hi[22] ^ S2_lo[43];
    S1_hxl[43] = S1_lo[58] ^ S1_lo[22] ^ S1_hi[22] ^ S2_lo[43];
    S2_hi[7] = S1_lo[58] ^ S1_lo[22] ^ S1_hi[22] ^ S2_lo[43];
    S2_hxl[7] = S1_lo[58] ^ S1_lo[7] ^ S1_hi[22] ^ S1_hi[7] ^ S2_lo[43];
    S1_cb[58] = ((S1_cb[57] & S1_is[41]) | (S1_is[41] & S1_lo[58]) | (S1_cb[57] & S1_lo[58]));
    S2_is[7] = S2_ca[6] ^ S2_hxl[7];
    S2_ca[7] = ((S2_ca[6] & S2_lo[7]) | (S2_lo[7] & S2_hi[7]) | (S2_ca[6] & S2_hi[7]));
    S2_ca[23] = ((S2_ca[22] & S2_lo[23]) | (S2_lo[23] & S2_hi[23]) | (S2_ca[22] & S2_hi[23]));
    S2_hxl[24] = S2_is[24] ^ S2_ca[23];
    S1_hi[24] = S1_lo[39] ^ S1_lo[24] ^ S1_lo[3] ^ S1_hi[3] ^ S1_hxl[60] ^ S2_hxl[24];
    S1_hxl[24] = S1_lo[39] ^ S1_lo[3] ^ S1_hi[3] ^ S1_hxl[60] ^ S2_hxl[24];
    S2_lo[24] = S1_hxl[60] ^ S2_hxl[24];
    S2_hi[52] = S1_lo[39] ^ S1_lo[3] ^ S1_hi[3] ^ S1_hxl[60] ^ S2_hxl[24];
    S2_hxl[52] = S1_lo[39] ^ S1_lo[31] ^ S1_hi[31] ^ S1_hi[3] ^ S1_hxl[60] ^ S1_hxl[52] ^ S2_hxl[24];
    S1_is[24] = S1_ca[23] ^ S1_hxl[24];
    S1_ca[24] = ((S1_ca[23] & S1_lo[24]) | (S1_lo[24] & S1_hi[24]) | (S1_ca[23] & S1_hi[24]));
    S1_lo[41] = S1_is[24] ^ S1_out[41] ^ S1_cb[40];
    S1_hi[41] = S1_lo[41] ^ S1_hxl[41];
    S2_lo[26] = S1_lo[41] ^ S1_lo[26] ^ S1_lo[5] ^ S1_hi[26] ^ S1_hi[5];
    S2_hxl[26] = S1_lo[41] ^ S1_lo[26] ^ S1_lo[5] ^ S1_hi[26] ^ S1_hi[5] ^ S1_hxl[62];
    S1_ca[41] = ((S1_ca[40] & S1_lo[41]) | (S1_lo[41] & S1_hi[41]) | (S1_ca[40] & S1_hi[41]));
    S1_is[42] = S1_ca[41] ^ S1_hxl[42];
    S1_cb[41] = ((S1_cb[40] & S1_is[24]) | (S1_is[24] & S1_lo[41]) | (S1_cb[40] & S1_lo[41]));
    S1_lo[59] = S1_is[42] ^ S1_out[59] ^ S1_cb[58];
    S1_hi[59] = S1_lo[59] ^ S1_hxl[59];
    S1_hi[44] = S1_lo[59] ^ S1_lo[44] ^ S1_lo[23] ^ S1_hi[23] ^ S2_lo[44];
    S1_hxl[44] = S1_lo[59] ^ S1_lo[23] ^ S1_hi[23] ^ S2_lo[44];
    S2_hi[8] = S1_lo[59] ^ S1_lo[23] ^ S1_hi[23] ^ S2_lo[44];
    S2_hxl[8] = S1_lo[59] ^ S1_lo[8] ^ S1_hi[23] ^ S1_hi[8] ^ S2_lo[44];
    S1_cb[59] = ((S1_cb[58] & S1_is[42]) | (S1_is[42] & S1_lo[59]) | (S1_cb[58] & S1_lo[59]));
    S2_is[8] = S2_ca[7] ^ S2_hxl[8];
    S2_ca[8] = ((S2_ca[7] & S2_lo[8]) | (S2_lo[8] & S2_hi[8]) | (S2_ca[7] & S2_hi[8]));
    S2_ca[24] = ((S2_ca[23] & S2_lo[24]) | (S2_lo[24] & S2_hi[24]) | (S2_ca[23] & S2_hi[24]));
    S2_hxl[25] = S2_is[25] ^ S2_ca[24];
    S1_hi[25] = S1_lo[40] ^ S1_lo[25] ^ S1_lo[4] ^ S1_hi[4] ^ S1_hxl[61] ^ S2_hxl[25];
    S1_hxl[25] = S1_lo[40] ^ S1_lo[4] ^ S1_hi[4] ^ S1_hxl[61] ^ S2_hxl[25];
    S2_lo[25] = S1_hxl[61] ^ S2_hxl[25];
    S2_hi[53] = S1_lo[40] ^ S1_lo[4] ^ S1_hi[4] ^ S1_hxl[61] ^ S2_hxl[25];
    S2_hxl[53] = S1_lo[53] ^ S1_lo[40] ^ S1_lo[32] ^ S1_hi[53] ^ S1_hi[32] ^ S1_hi[4] ^ S1_hxl[61] ^ S2_hxl[25];
    S1_is[25] = S1_ca[24] ^ S1_hxl[25];
    S1_lo[42] = S1_is[25] ^ S1_out[42] ^ S1_cb[41];
    S1_hi[42] = S1_lo[42] ^ S1_hxl[42];
    S2_lo[27] = S1_lo[42] ^ S1_lo[27] ^ S1_lo[6] ^ S1_hi[27] ^ S1_hi[6];
    S2_hxl[27] = S1_lo[42] ^ S1_lo[27] ^ S1_lo[6] ^ S1_hi[27] ^ S1_hi[6] ^ S1_hxl[63];
    S1_ca[42] = ((S1_ca[41] & S1_lo[42]) | (S1_lo[42] & S1_hi[42]) | (S1_ca[41] & S1_hi[42]));
    S1_is[43] = S1_ca[42] ^ S1_hxl[43];
    S1_ca[43] = ((S1_ca[42] & S1_lo[43]) | (S1_lo[43] & S1_hi[43]) | (S1_ca[42] & S1_hi[43]));
    S1_is[44] = S1_ca[43] ^ S1_hxl[44];
    S1_ca[44] = ((S1_ca[43] & S1_lo[44]) | (S1_lo[44] & S1_hi[44]) | (S1_ca[43] & S1_hi[44]));
    S1_lo[60] = S1_is[43] ^ S1_out[60] ^ S1_cb[59];
    S1_hi[60] = S1_lo[60] ^ S1_hxl[60];
    S1_hi[45] = S1_lo[60] ^ S1_lo[45] ^ S1_lo[24] ^ S1_hi[24] ^ S2_lo[45];
    S1_hxl[45] = S1_lo[60] ^ S1_lo[24] ^ S1_hi[24] ^ S2_lo[45];
    S2_hi[9] = S1_lo[60] ^ S1_lo[24] ^ S1_hi[24] ^ S2_lo[45];
    S2_hxl[9] = S1_lo[60] ^ S1_lo[9] ^ S1_hi[24] ^ S1_hi[9] ^ S2_lo[45];
    S1_is[45] = S1_ca[44] ^ S1_hxl[45];
    S1_ca[45] = ((S1_ca[44] & S1_lo[45]) | (S1_lo[45] & S1_hi[45]) | (S1_ca[44] & S1_hi[45]));
    S1_cb[60] = ((S1_cb[59] & S1_is[43]) | (S1_is[43] & S1_lo[60]) | (S1_cb[59] & S1_lo[60]));
    S1_lo[61] = S1_is[44] ^ S1_out[61] ^ S1_cb[60];
    S1_hi[61] = S1_lo[61] ^ S1_hxl[61];
    S1_hi[46] = S1_lo[61] ^ S1_lo[46] ^ S1_lo[25] ^ S1_hi[25] ^ S2_lo[46];
    S1_hxl[46] = S1_lo[61] ^ S1_lo[25] ^ S1_hi[25] ^ S2_lo[46];
    S2_hi[10] = S1_lo[61] ^ S1_lo[25] ^ S1_hi[25] ^ S2_lo[46];
    S2_hxl[10] = S1_lo[61] ^ S1_lo[10] ^ S1_hi[25] ^ S1_hi[10] ^ S2_lo[46];
    S1_is[46] = S1_ca[45] ^ S1_hxl[46];
    S1_cb[61] = ((S1_cb[60] & S1_is[44]) | (S1_is[44] & S1_lo[61]) | (S1_cb[60] & S1_lo[61]));
    S1_cb[62] = ((S1_cb[61] & S1_is[45]) | (S1_is[45] & S1_lo[62]) | (S1_cb[61] & S1_lo[62]));
    S1_cb[63] = ((S1_cb[62] & S1_is[46]) | (S1_is[46] & S1_lo[63]) | (S1_cb[62] & S1_lo[63]));
    S2_is[9] = S2_ca[8] ^ S2_hxl[9];
    S2_ca[9] = ((S2_ca[8] & S2_lo[9]) | (S2_lo[9] & S2_hi[9]) | (S2_ca[8] & S2_hi[9]));
    S2_is[10] = S2_ca[9] ^ S2_hxl[10];
    S2_ca[25] = ((S2_ca[24] & S2_lo[25]) | (S2_lo[25] & S2_hi[25]) | (S2_ca[24] & S2_hi[25]));
    S2_ca[26] = ((S2_ca[25] & S2_lo[26]) | (S2_lo[26] & S2_hi[26]) | (S2_ca[25] & S2_hi[26]));
    S2_ca[29] = ((S2_ca[28] & S2_lo[29]) | (S2_lo[29] & S2_hi[29]) | (S2_ca[28] & S2_hi[29]));
    S2_is[30] = S2_ca[29] ^ S2_hxl[30];
    S2_ca[30] = ((S2_ca[29] & S2_lo[30]) | (S2_lo[30] & S2_hi[30]) | (S2_ca[29] & S2_hi[30]));
    S2_is[31] = S2_ca[30] ^ S2_hxl[31];
    S2_is[50] = S2_ca[49] ^ S2_hxl[50];
    S2_ca[50] = ((S2_ca[49] & S2_lo[50]) | (S2_lo[50] & S2_hi[50]) | (S2_ca[49] & S2_hi[50]));
    S2_is[51] = S2_ca[50] ^ S2_hxl[51];
    S2_ca[51] = ((S2_ca[50] & S2_lo[51]) | (S2_lo[51] & S2_hi[51]) | (S2_ca[50] & S2_hi[51]));
    S2_is[52] = S2_ca[51] ^ S2_hxl[52];
    S2_ca[52] = ((S2_ca[51] & S2_lo[52]) | (S2_lo[52] & S2_hi[52]) | (S2_ca[51] & S2_hi[52]));
    S2_cb[3] = ((S2_cb[2] & S2_is[50]) | (S2_is[50] & S2_lo[3]) | (S2_cb[2] & S2_lo[3]));
    S2_cb[4] = ((S2_cb[3] & S2_is[51]) | (S2_is[51] & S2_lo[4]) | (S2_cb[3] & S2_lo[4]));
    S2_cb[21] = ((S2_cb[20] & S2_is[4]) | (S2_is[4] & S2_lo[21]) | (S2_cb[20] & S2_lo[21]));
    S2_cb[22] = ((S2_cb[21] & S2_is[5]) | (S2_is[5] & S2_lo[22]) | (S2_cb[21] & S2_lo[22]));
    S2_cb[23] = ((S2_cb[22] & S2_is[6]) | (S2_is[6] & S2_lo[23]) | (S2_cb[22] & S2_lo[23]));
    S2_cb[24] = ((S2_cb[23] & S2_is[7]) | (S2_is[7] & S2_lo[24]) | (S2_cb[23] & S2_lo[24]));
    S2_cb[25] = ((S2_cb[24] & S2_is[8]) | (S2_is[8] & S2_lo[25]) | (S2_cb[24] & S2_lo[25]));
    S2_cb[26] = ((S2_cb[25] & S2_is[9]) | (S2_is[9] & S2_lo[26]) | (S2_cb[25] & S2_lo[26]));
    S2_cb[46] = ((S2_cb[45] & S2_is[29]) | (S2_is[29] & S2_lo[46]) | (S2_cb[45] & S2_lo[46]));
    S2_cb[47] = ((S2_cb[46] & S2_is[30]) | (S2_is[30] & S2_lo[47]) | (S2_cb[46] & S2_lo[47]));
    S2_cb[63] = ((S2_cb[62] & S2_is[46]) | (S2_is[46] & S2_lo[63]) | (S2_cb[62] & S2_lo[63]));
    //  END   ▲▲▲▲▲
    // ----------------------------------------------------------------------------

    // ----------------------------------------------------------------------------
    //  4.   Assemble the two 64-bit results from bit slices
    // ----------------------------------------------------------------------------
    uint64_t result_lo = 0ULL;
    uint64_t result_hi = 0ULL;
    
    #pragma unroll
    for (int i = 0; i < 64; ++i) {
        result_lo |= (static_cast<uint64_t>(S1_lo[i]) & 1ULL) << i;
        result_hi |= (static_cast<uint64_t>(S1_hi[i]) & 1ULL) << i;
    }

    // ----------------------------------------------------------------------------
    //  5.   Validate; if good, push to results buffer via atomic index
    // ----------------------------------------------------------------------------
    if (isValid(result_lo, result_hi, nextlong1, nextlong2))
    {
        unsigned int idx = atomicInc(d_resIndex, RESULTS_BUFFER_SIZE - 1);
        if (idx < RESULTS_BUFFER_SIZE)    // simple overflow protection
        {
            d_results[idx].guess_bits = guess & ((1ULL << 42) - 1);
            d_results[idx].result_lo  = result_lo;
            d_results[idx].result_hi  = result_hi;
        }
    }
}

// ──────────────────────────────────────────────────────────────────────
//  Host-side helpers
// ──────────────────────────────────────────────────────────────────────
static void usage(const char *prog)
{
    std::cerr
        << "Usage: " << prog
        << " -nl1 <nextlong1> -nl2 <nextlong2> [-d <cudaDevice>] "
        << "[-s <startGuess>] [-e <endGuess>]\n";
}

static unsigned long long atoull(const char *s)
{
    char *end = nullptr;
    unsigned long long v = std::strtoull(s, &end, 0);
    if (end == s || *end != '\0') {
        std::cerr << "Invalid numeric argument: " << s << std::endl;
        std::exit(EXIT_FAILURE);
    }
    return v;
}

// ──────────────────────────────────────────────────────────────────────
//  main()
// ──────────────────────────────────────────────────────────────────────
int main(int argc, char *argv[])
{
    if (argc < 5) {
        usage(argv[0]);
        return EXIT_FAILURE;
    }

    int deviceId = 0;
    uint64_t nextlong1 = 0, nextlong2 = 0;
    unsigned long long startGuess = 0, endGuess = (1ULL << 42);   // full space

    // Very light-weight argument parse
    for (int i = 1; i < argc; ++i) {
        if (!std::strcmp(argv[i], "-nl1") && i + 1 < argc) {
            nextlong1 = atoull(argv[++i]);
        } else if (!std::strcmp(argv[i], "-nl2") && i + 1 < argc) {
            nextlong2 = atoull(argv[++i]);
        } else if (!std::strcmp(argv[i], "-d") && i + 1 < argc) {
            deviceId = std::atoi(argv[++i]);
        } else if (!std::strcmp(argv[i], "-s") && i + 1 < argc) {
            startGuess = atoull(argv[++i]);
        } else if (!std::strcmp(argv[i], "-e") && i + 1 < argc) {
            endGuess = atoull(argv[++i]);
        } else {
            usage(argv[0]);
            return EXIT_FAILURE;
        }
    }

    // -----------------------------------------------------------------
    //  Select GPU and report
    // -----------------------------------------------------------------
    CUDA_CHECK(hipSetDevice(deviceId));
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, deviceId));
    std::cout << "Running on GPU " << deviceId << " ["
              << prop.name << "]\n";

    // -----------------------------------------------------------------
    //  Allocate device buffers
    // -----------------------------------------------------------------
    Result *d_results = nullptr;
    unsigned int *d_resIndex = nullptr;
    CUDA_CHECK(hipMalloc(&d_results, RESULTS_BUFFER_SIZE * sizeof(Result)));
    CUDA_CHECK(hipMalloc(&d_resIndex, sizeof(unsigned int)));

    // Host-side result scratch
    std::vector<Result> h_results(RESULTS_BUFFER_SIZE);

    // -----------------------------------------------------------------
    //  Iterate batches
    // -----------------------------------------------------------------
    unsigned long long totalTested = 0;
    auto t_start = std::chrono::high_resolution_clock::now();

    for (unsigned long long batchBase = startGuess;
         batchBase < endGuess;
         batchBase += BATCH_SIZE)
    {
        // Reset result index to 0
        unsigned int zero = 0;
        CUDA_CHECK(hipMemcpy(d_resIndex, &zero, sizeof(unsigned int),
                              hipMemcpyHostToDevice));

        // Launch grid
        dim3 grid(BLOCK_SIZE);
        dim3 block(THREAD_SIZE);

        bruteKernel<<<grid, block>>>(nextlong1, nextlong2, batchBase,
                                     d_results, d_resIndex);
        CUDA_CHECK(hipGetLastError());

        // Copy back results index
        unsigned int h_resCount = 0;
        CUDA_CHECK(hipMemcpy(&h_resCount, d_resIndex, sizeof(unsigned int),
                              hipMemcpyDeviceToHost));

        if (h_resCount > RESULTS_BUFFER_SIZE)
            h_resCount = RESULTS_BUFFER_SIZE;

        if (h_resCount) {
            CUDA_CHECK(hipMemcpy(h_results.data(), d_results,
                                  h_resCount * sizeof(Result),
                                  hipMemcpyDeviceToHost));

            std::cout << '\n';
            for (unsigned int i = 0; i < h_resCount; ++i) {
                const auto &r = h_results[i];
                std::cout << "FOUND - guess 0x" << std::hex << r.guess_bits
                          << "  result_lo:0x" << r.result_lo
                          << "  result_hi:0x" << r.result_hi << std::dec << '\n';
            }
        }

        totalTested += BATCH_SIZE;
        auto now = std::chrono::high_resolution_clock::now();
        double elapsedSec = std::chrono::duration<double>(now - t_start).count();
        double MHs = (totalTested / 1e6) / elapsedSec;

        std::cout << "\rBatch complete. Tested "
                  << std::setw(10) << totalTested
                  << " inputs [" << std::fixed << std::setprecision(2)
                  << MHs << " MH/s]" << std::flush;
    }

    std::cout << "\nAll batches finished.\n";

    // -----------------------------------------------------------------
    //  Cleanup
    // -----------------------------------------------------------------
    CUDA_CHECK(hipFree(d_results));
    CUDA_CHECK(hipFree(d_resIndex));
    return 0;
}
